
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void hello_world()
{
 printf("Hello World From GPU!\n");
}

int main(void)
{
  hello_world<<<1, 1>>>();
  hipDeviceReset();
  return 0;
}
