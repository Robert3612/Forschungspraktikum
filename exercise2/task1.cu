
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <iostream>

template <typename T>
__global__ void add(T* A, T* B, T* C,size_t elementcount) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    for(size_t i=0 + id*16;i<=15+id*16; i++)
    {
        C[i] = A[i] + B[i];
    }
}

template <typename T>
void add_cpu(T* A, T* B, T* C, size_t elementcount) {
    for (size_t i = 0; i < (int) elementcount; i++) {
        C[i] = A[i] + B[i];
    }
}

template <typename T>
void validate(T* h, T* d, size_t elementcount) {
    for (size_t i = 0; i < elementcount; i++) {
        if (h[i] != d[i]) {
            std::cout << "found invalidated field in element " << i << std::endl;
            std::cout << "on CPU side: " << h[i] << std::endl;
            std::cout << "on GPU side: " << d[i] << std::endl;
        }
    }
}

int main(void)
{
    size_t elementcount=1048576;
    uint64_t* h_A;
    uint64_t* h_B;
    uint64_t* h_C;

    uint64_t* h_out;

    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t bytes = elementcount * sizeof(uint64_t);

    h_A = (uint64_t*)malloc(bytes);
    h_B = (uint64_t*)malloc(bytes);
    h_C = (uint64_t*)malloc(bytes);
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);


    for (size_t i = 0; i < elementcount; i++) {
        h_A[i] = rand() % 8000 + 1;
        h_B[i] = rand() % 8000 + 1;
    }

        hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, bytes);
	
    hipEventRecord(start);
    add <<<64, 1024 >>> (d_A, d_B, d_C, elementcount);
    hipEventRecord(stop);
    
    hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout<<"Time of kernel in milliseconds: "<< milliseconds<< "ms" << std::endl;

    add_cpu(h_A, h_B, h_C, elementcount);


    validate(h_C, h_out, elementcount);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    free(h_out);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
