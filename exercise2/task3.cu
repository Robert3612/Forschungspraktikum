
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>

template <typename T>
__global__ void add(T* A, T* B, T* C,size_t elementcount) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        C[i] = A[i] + B[i];
    }
}

template <typename T>
void add_cpu(T* A, T* B, T* C, size_t elementcount) {
    for (size_t i = 0; i < (int) elementcount; i++) {
        C[i] = A[i] + B[i];
    }
}

template <typename T>
void validate(T* h, T* d, size_t elementcount) {
    for (size_t i = 0; i < elementcount; i++) {
        if (h[i] != d[i]) {
            std::cout << "found invalidated field in element " << i << std::endl;
            std::cout << "on CPU side: " << h[i] << std::endl;
            std::cout << "on GPU side: " << d[i] << std::endl;
        }
    }
}

int main(void)
{
    size_t elementcount=1048576;
    uint64_t* h_A;
    uint64_t* h_B;
    uint64_t* h_C;

    uint64_t* h_out;

    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;

    std::ofstream myFile("add.csv");

// for appending
//    std::ofstream myFile("add.csv", std::ios::app);
//    myFile << "\n" << "\n";

    myFile << "kernel;element_count;block_count;thread_count;time_ms;throughput\n";

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t bytes = elementcount * sizeof(uint64_t);

    h_A = (uint64_t*)malloc(bytes);
    h_B = (uint64_t*)malloc(bytes);
    h_C = (uint64_t*)malloc(bytes);
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);


    for (size_t i = 0; i < elementcount; i++) {
        h_A[i] = rand() % 8000 + 1;
        h_B[i] = rand() % 8000 + 1;
    }

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    for(int i=16;i<=1024;i=i*2){
        for(int j=8;j<=512;j=j*2){
            hipMemset(d_C, 0, bytes);
	
            hipEventRecord(start);
            add <<<j, i >>> (d_A, d_B, d_C, elementcount);
            hipEventRecord(stop);
    
            hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            add_cpu(h_A, h_B, h_C, elementcount);


            validate(h_C, h_out, elementcount);

            myFile << "add" << ";";
            myFile << elementcount << ";";
            myFile << j << ";";
            myFile << i << ";";
            myFile << milliseconds << ";";
            myFile << elementcount*8*2/milliseconds/1e6 << "\n";
        }
    }



    myFile.close();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    free(h_out);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
