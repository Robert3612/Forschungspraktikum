
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <iostream>

template <typename T>
__global__ void aggregate_shared(T* A, T* C,size_t elementcount) {
    extern __shared__ uint64_t temp[];
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step;
    if(elementcount > blockDim.x * gridDim.x){
        step = blockDim.x * gridDim.x;
    }
    else{
        step = elementcount/2;
    }
    int border = elementcount;
    for (size_t i = id; i < border; i += step){
        temp[i] = A[i];
    
    }
    while(border> 1){
    if(id <= step){
    for (size_t i = id; i < border; i += step){ 
        if(id != i){
        temp[id] = temp[id] + temp[i];
        }
    }
    border = step;
    step = step/2;
    }
    else{
        break;
    }
    __syncthreads();
    }
    if(id==0){
        *C=temp[0];
    }
}

template <typename T>
__global__ void aggregate(T* A, T* C,size_t elementcount) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step;
    if(elementcount > blockDim.x * gridDim.x){
        step = blockDim.x * gridDim.x;
    }
    else{
        step = elementcount/2;
    }
    int border = elementcount;
    
    while(border> 1){
    if(id <= step){
    for (size_t i = id; i < border; i += step){ 
        if(id != i){
        A[id] = A[id] + A[i];
        }
    }
    border = step;
    step = step/2;
    }
    else{
        break;
    }
    __syncthreads();
    }
    if(id==0){
        *C=A[0];
    }
}

template <typename T>
void aggregate_cpu(T* A, T* C, size_t elementcount) {
    for (size_t i = 0; i < (int) elementcount; i++) {
        *C = *C + A[i];
    }
}

template <typename T>
void validate(T* h, T* d) {

        if (*h != *d) {
            std::cout << "found invalidated answer" << std::endl;
            std::cout << "on CPU side: " << *h << std::endl;
            std::cout << "on GPU side: " << *d << std::endl;
        }
    
}

int main(void)
{
    size_t elementcount=10;
    size_t bytes = elementcount * sizeof(uint64_t);
    uint64_t* h_A;
    uint64_t* h_C;

    uint64_t* h_out;

    uint64_t* d_A;
    uint64_t* d_C;



    h_A = (uint64_t*)malloc(bytes);
    h_C = (uint64_t*)malloc(sizeof(uint64_t));
    h_out = (uint64_t*)malloc(sizeof(uint64_t));

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_C, sizeof(uint64_t));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    for (size_t i = 0; i < elementcount; i++) {
        h_A[i] = rand() % 8000 + 1;
    }

    aggregate_cpu(h_A, h_C, elementcount);

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, sizeof(uint64_t));
	
    hipEventRecord(start2);
    aggregate <<<1, 4 >>> (d_A, d_C, elementcount);
    hipEventRecord(stop2);

    hipEventSynchronize(stop2);

    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start2, stop2);

    std::cout<<"Time of kernel in milliseconds: "<< milliseconds2<< "ms" << std::endl;
    
    hipMemcpy(h_out, d_C, sizeof(uint64_t), hipMemcpyDeviceToHost);

    validate(h_C, h_out);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, sizeof(uint64_t));
	
    hipEventRecord(start);
    aggregate_shared <<<1, 4, elementcount*sizeof(uint64_t) >>> (d_A, d_C, elementcount);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout<<"Time of kernel in milliseconds (shared_memory): "<< milliseconds<< "ms" << std::endl;

    hipMemcpy(h_out, d_C, sizeof(uint64_t), hipMemcpyDeviceToHost);

    validate(h_C, h_out);


    hipFree(d_A);
    hipFree(d_C);


    free(h_out);
    free(h_A);
    free(h_C);

    return 0;


    //comparison:
    //shared memory is faster than non-shared method
}
