
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>

std::string decode(std::string s);
int unBinary_test(std::string code);
std::string binary_test(int apperance);

template <typename T>
using EnableIfIntegral = std::enable_if_t<std::is_integral<T>::value>;


template <size_t T>
class Slab
{
private:
    std::bitset<T> bitset;
    std::vector<int> valueBits;

public:
    template <size_t E>
    static std::vector<Slab<E>> packBitsets(const std::vector<std::string> &values)
    {

        auto is_too_big = [](std::string i)
        { return i.length() > T; };

        if (auto h = std::find_if(values.begin(), values.end(), is_too_big); h != values.end())
        {
            std::cout << "Error: Integers are bigger than bitset" << std::endl;
            exit(EXIT_FAILURE);
        }

        int bit_position = 0;
        auto slabs = std::vector<Slab<E>>();

        auto slab = Slab<E>();

        for (auto s : values)
        {
            if ((bit_position + s.length()) > T)
            {
                slabs.push_back(slab);
                slab = Slab<E>();
            }

            for (auto &ch : s)
            {
                slab.bitset.set(bit_position, ch == '1');
                bit_position++;
            }

            slab.valueBits.push_back(s.length());
        }

        slabs.push_back(slab);

        return slabs;
    }

    std::vector<uint64_t> unpack()
    {

        auto output = std::vector<uint64_t>();

        int start_bit = 0;
        std::string outputS;
        for (auto end_bit : valueBits)
        {
            std::string s;

            for (auto i = start_bit; i < end_bit + start_bit; i++)
            {
                auto h = bitset[i];
                if (h)
                    s.push_back('1');
                else
                    s.push_back('0');
            }
            outputS = decode(s);
            output.push_back(std::stoi(outputS, nullptr, 2));
            start_bit = end_bit;
        }

        return output;
    }
};


std::string runLengthEliasGamma(std::vector<std::string> &vector, std::vector<int> &mask)
{   std::string output = "";
    for (auto i = 0; i < vector.size(); i++)
    {
        
        char letter = vector[i][0];
        int apperance = 1;
        std::string newString = "";
        std::string helper;
        int mask_helper = 0;
        for(int j=1;j<vector[i].length();j++) {
            if(letter != vector[i][j]){
                if(apperance == 1){
                mask_helper = mask_helper + 1;
                mask.push_back(mask_helper);
                newString = newString + letter;    
                letter = vector[i][j];    
                }
                else{
                 helper = binary_test(apperance);
                // std::cout<< "helper" + helper<< std::endl;
                // std::cout<< "helper_length"<< helper.length() << std::endl;
                mask_helper = mask_helper + helper.length() + 1;
                //std::cout<< mask_helper << std::endl; 
                mask.push_back(mask_helper);
                newString = newString + letter + helper;
                letter = vector[i][j];
                apperance=1;   
                }
            }
            else{
                apperance++;
            }
        }
        if(apperance == 1){
                newString = newString + letter;
                mask_helper = mask_helper ;
                mask.push_back(mask_helper);
                output = output + newString;  
                vector[i] = newString; 
                }
                else{
        helper = binary_test(apperance);
        newString = newString + letter + helper;
        mask_helper = mask_helper + helper.length() ;
        mask.push_back(mask_helper);
        output = output + newString;
        vector[i] = newString;
}
    }
    return output;
}

int unElias(std::string code){
    if(code != ""){
    int length = code.length();
    int answer = pow(2, length);
    answer = answer + std::stoi(code, nullptr, 2);
    return answer;
    }
    return 1;
}

std::string decode(std::string s, std::vector<int> mask){
    std::string newString = "";
    std::string helper;
    char letter;
    int app_helper;
    if(mask.at(0)-1 ==0){
        newString = newString + s[0];
    }
    else{
        helper = s.substr(1,mask.at(0)-1);
        letter = s[0]; 
        app_helper = unBinary_test(helper);
        newString= newString + std::string(app_helper, letter);
    }
    //std::cout<< newString<< std::endl;
    
    for(int i=0;i<mask.size()-1;i++){
    if(mask.at(i)< mask.at(i+1)){
        
    }
    else{
        if(mask.at(i+1)-1 == mask.at(i)){
        newString = newString + s[mask.at(i)];
    }
    else{
        helper = s.substr(mask.at(i)+1,(mask.at(i+1)-1)-mask.at(i));
        letter = s[mask.at(i)]; 
        app_helper = unBinary_test(helper);
        newString= newString + std::string(app_helper, letter);
    }
    }
    }
    
    if(mask.back() == s.length()-1){
        newString = newString + s[mask.back()];
    }
    else{
        helper = s.substr(mask.back(),(s.length()-1)-mask.back());
        letter = s[mask.back()]; 
        app_helper = unBinary_test(helper);
        newString= newString + std::string(app_helper, letter);
    }
    return newString;
}

std::string binary_test(int apperance)
{
std::string binary = std::bitset<8>(apperance).to_string();
binary.erase(0, binary.find_first_not_of('0'));

//std::cout<< binary<<std::endl;
return binary;
}

int unBinary_test(std::string code){
    int answer = std::stoi(code, nullptr, 2);
    //std::cout<< answer << std::endl;
    return answer;
}

int main()
{
    std::vector<std::string> h{"100110100001010011", "000110100001"};
    for (auto s : h)
        std::cout << s << std::endl;
    std::vector<int> mask;
    std::string decoded = runLengthEliasGamma(h, mask);
    
    for (auto s : h)
        std::cout << s << std::endl;
    std::cout<< decoded <<std::endl;
        
    for (auto i: mask)
        std::cout << i << std::endl;
    
    //std::cout<< decode(h.at(0), mask)<<std::endl;
    
    
    
}