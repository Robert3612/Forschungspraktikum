
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>

template <typename T>
using EnableIfIntegral = std::enable_if_t<std::is_integral<T>::value>;

template <size_t T>
class Slab
{
private:
    std::bitset<T> bitset;
    std::vector<int> valueBits;

public:
    template <size_t E>
    static std::vector<Slab<E>> packBitsets(const std::vector<std::string> &values)
    {

        auto is_too_big = [](std::string i)
        { return i.length() > T; };

        if (auto h = std::find_if(values.begin(), values.end(), is_too_big); h != values.end())
        {
            std::cout << "Error: Integers are bigger than bitset" << std::endl;
            exit(EXIT_FAILURE);
        }

        int bit_position = 0;
        auto slabs = std::vector<Slab<E>>();

        auto slab = Slab<E>();

        for (auto s : values)
        {
            if ((bit_position + s.length()) > T)
            {
                slabs.push_back(slab);
                slab = Slab<E>();
            }

            for (auto &ch : s)
            {
                slab.bitset.set(bit_position, ch == '1');
                bit_position++;
            }

            slab.valueBits.push_back(s.length());
        }

        slabs.push_back(slab);

        return slabs;
    }

    std::vector<uint64_t> unpack()
    {

        auto output = std::vector<uint64_t>();

        int start_bit = 0;

        for (auto end_bit : valueBits)
        {
            std::string s;

            for (auto i = start_bit; i < end_bit + start_bit; i++)
            {
                auto h = bitset[i];
                if (h)
                    s.push_back('1');
                else
                    s.push_back('0');
            }

            output.push_back(std::stoi(s, nullptr, 2));
            start_bit = end_bit;
        }

        return output;
    }
};

void removeLeadingZeros(std::vector<std::string> &vector)
{

    for (auto i = 0; i < vector.size(); i++)
    {
        auto pos = vector[i].find('1');
        if (pos > 0)
            vector[i].erase(0, pos);
    }
}

int main()
{
    // 2465,417
    std::vector<std::string> h{"100110100001", "000110100001"};

    removeLeadingZeros(h);

    auto slabs = Slab<64>::packBitsets<64>(h);

    auto slab = slabs.front();

    for (auto s : slab.unpack())
        std::cout << s << std::endl;
}