
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>

std::string decode(std::string s);
int unBinary_test(std::string code);
std::string binary_test(int apperance);

template <typename T>
__global__ void decode(T* code, T* outcome,T* mask1, T* mask2, size_t elementcount1, size_t elementcount2) {
    int pos1;
    int pos2;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i <= elementcount2/3){
        pos1 = mask2[i*3];
        pos2 = mask2[i*3+2];
        
    }
}


template <typename T>
using EnableIfIntegral = std::enable_if_t<std::is_integral<T>::value>;


template <size_t T>
class Slab
{
private:
    std::bitset<T> bitset;
    std::vector<int> valueBits;

public:
    template <size_t E>
    static std::vector<Slab<E>> packBitsets(const std::vector<std::string> &values)
    {

        auto is_too_big = [](std::string i)
        { return i.length() > T; };

        if (auto h = std::find_if(values.begin(), values.end(), is_too_big); h != values.end())
        {
            std::cout << "Error: Integers are bigger than bitset" << std::endl;
            exit(EXIT_FAILURE);
        }

        int bit_position = 0;
        auto slabs = std::vector<Slab<E>>();

        auto slab = Slab<E>();

        for (auto s : values)
        {
            if ((bit_position + s.length()) > T)
            {
                slabs.push_back(slab);
                slab = Slab<E>();
            }

            for (auto &ch : s)
            {
                slab.bitset.set(bit_position, ch == '1');
                bit_position++;
            }

            slab.valueBits.push_back(s.length());
        }

        slabs.push_back(slab);

        return slabs;
    }

    std::vector<uint64_t> unpack()
    {

        auto output = std::vector<uint64_t>();

        int start_bit = 0;
        std::string outputS;
        for (auto end_bit : valueBits)
        {
            std::string s;

            for (auto i = start_bit; i < end_bit + start_bit; i++)
            {
                auto h = bitset[i];
                if (h)
                    s.push_back('1');
                else
                    s.push_back('0');
            }
            outputS = decode(s);
            output.push_back(std::stoi(outputS, nullptr, 2));
            start_bit = end_bit;
        }

        return output;
    }
};


std::string runLengthEliasGamma(std::vector<std::string> &vector, std::vector<int> &mask, std::vector<int> &mask2)
{   std::string decoded = "";
    std::string output = "";
    int back;
    std::vector<int> helper_vector;
    helper_vector.push_back(vector[0].size()-1);
    decoded = decoded + vector[0];
    for (auto i = 1; i < vector.size(); i++)
    {
        helper_vector.push_back(helper_vector.back() + vector[i].size());
        decoded = decoded + vector[i];
    }
    
        
        char letter = decoded[0];
        int apperance = 1;
        std::string newString = "";
        std::string helper;
        int mask_helper = 0;
        int index_helper = 0;
        bool after_index = false;
        for(int j=1;j<decoded.length();j++) {
            if(letter != decoded[j]){
                if(apperance == 1){
                mask_helper = mask_helper + 1;
                mask2.push_back(mask_helper);
                newString = newString + letter;    
                letter = decoded[j];  
                }
                else{
                 helper = binary_test(apperance);
                mask_helper = mask_helper + helper.length() + 1;
                mask2.push_back(mask_helper);
                newString = newString + letter + helper;
                letter = decoded[j];
                apperance=1;   
                }
            if(after_index){
                mask.push_back(newString.length()-1);
                after_index = false;
            }
            if(helper_vector.at(index_helper)==j){
                    mask.push_back(newString.length());
                    mask.push_back(apperance);
                    index_helper++;
                    after_index = true;
            }
            }
            else{
                apperance++;
                if(helper_vector.at(index_helper)==j){
                    mask.push_back(newString.length());
                    mask.push_back(apperance);
                    index_helper++;
                    after_index = true;
            }
            }
        }

        if(apperance == 1){
                newString = newString + letter;
                mask_helper = mask_helper ;
                mask2.push_back(mask_helper);
                output = output + newString;  
                }
                else{
        helper = binary_test(apperance);
        newString = newString + letter + helper;
        mask_helper = mask_helper + helper.length() ;
        mask2.push_back(mask_helper);
        output = output + newString;
        }
        mask.push_back(newString.length()-1);
        
    
    return output;
}

int unElias(std::string code){
    if(code != ""){
    int length = code.length();
    int answer = pow(2, length);
    answer = answer + std::stoi(code, nullptr, 2);
    return answer;
    }
    return 1;
}

std::string decode(std::string s, std::vector<int> mask){
    std::string newString = "";
    std::string helper;
    char letter;
    int app_helper;
    if(mask.at(0)-1 ==0){
        newString = newString + s[0];
    }
    else{
        helper = s.substr(1,mask.at(0)-1);
        letter = s[0]; 
        app_helper = unBinary_test(helper);
        newString= newString + std::string(app_helper, letter);
    }
    //std::cout<< newString<< std::endl;
    
    for(int i=0;i<mask.size()-1;i++){
    if(mask.at(i)< mask.at(i+1)){
        
    }
    else{
        if(mask.at(i+1)-1 == mask.at(i)){
        newString = newString + s[mask.at(i)];
    }
    else{
        helper = s.substr(mask.at(i)+1,(mask.at(i+1)-1)-mask.at(i));
        letter = s[mask.at(i)]; 
        app_helper = unBinary_test(helper);
        newString= newString + std::string(app_helper, letter);
    }
    }
    }
    
    if(mask.back() == s.length()-1){
        newString = newString + s[mask.back()];
    }
    else{
        helper = s.substr(mask.back(),(s.length()-1)-mask.back());
        letter = s[mask.back()]; 
        app_helper = unBinary_test(helper);
        newString= newString + std::string(app_helper, letter);
    }
    return newString;
}

std::string binary_test(int apperance)
{
std::string binary = std::bitset<10>(apperance).to_string();
binary.erase(0, binary.find_first_not_of('0'));

//std::cout<< binary<<std::endl;
return binary;
}

int unBinary_test(std::string code){
    int answer = std::stoi(code, nullptr, 2);
    //std::cout<< answer << std::endl;
    return answer;
}

void generate(std::vector<int> &start){
    for(int i=0;i<64;i++){
        start.push_back(rand() % 1024);
    }
}

std::vector<std::string> intToBinary(std::vector<int> start, std::string &outcome){
    std::string helper = "";
    std::vector<std::string> h;
    for(int i=0;i<start.size();i++){
        helper = binary_test(start.at(i));
        outcome = outcome + helper;
        h.push_back(helper);
    }
    
    return h;
}

int main()
{
    std::vector<int> start;
    std::string outcome;
    
    generate(start);
    for (auto i: start)
        std::cout << i << ", ";
    std::vector<std::string> h;
    
    h = intToBinary(start, outcome);
    for (auto s : h)
        std::cout << s << std::endl;
    std::cout<< outcome <<std::endl;
    std::vector<int> mask;
    std::vector<int> mask2;
    std::string decoded = runLengthEliasGamma(h, mask, mask2);
    
    
    //for (auto s : h)
    //    std::cout << s << std::endl;
    std::cout<< decoded <<std::endl;
        
    for (auto i: mask)
        std::cout << i << ", ";
    std::cout << std::endl;    
    for (auto i: mask2)
        std::cout << i << ", ";
    
    //std::cout<< decode(h.at(0), mask)<<std::endl;
    
    
    
}