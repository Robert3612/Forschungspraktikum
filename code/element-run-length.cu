#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>

   __device__ int decode_int(int *array, int* mask, int i)
{
    int helper;
    int length;
    int char_length;
    int pre_length;
    char helper_array[10];
    int answer = 0;
    int stelle = 0;

    helper = mask_A[i*3];
    int number = A[helper];
    char_length = A[helper+1];
    length = mask_A[i*3+1];
    pre_length = mask_A[i*3-2] - mask_A[i*3-1];
    if(length == mask_A[i*3+2]){
        for(int i=0;i< length;i++){
            answer = answer + number * pow(10,i);
        }
    }else{
        if(pre_length > 0){
            char_length = char_length - pre_length;
        }
        while(length> 0){
            answer = answer + number * pow(10, stelle);


        }
    }

    return answer;
}




__global__ 
void add(int *A, int *B, int *C, int *mask_A, int *mask_B,int length_A, int length_B, int elementcount) {
    int a;
    int b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int(A, mask_A, i);
        b = decode_int(B, mask_B, i);

        C[i] = a + b;
    }
    __syncthreads();
    if(id == 0){
    encode(C);
    }
}


__global__
void hello_world()
{
 printf("Hello World From GPU!\n");
}

void encode(std::vector<int> start, std::vector<int> &mask, std::string &outcome){
    std::string input = std::to_string(start.at(0));
    std::vector<int> length;
    length.push_back(std::to_string(start.at(0)).length());
    std::vector<int> pos;
    pos.push_back(std::to_string(start.at(0)).size()-1);
    for(int i=1;i<start.size();i++){
        input = input + std::to_string(start.at(i));
        length.push_back(std::to_string(start.at(i)).length());
        pos.push_back(pos.back() + std::to_string(start.at(i)).length());
    }
    
    char letter = input[0];
    int apperance = 1;
    int length_index = 0;
    int length_helper = length.at(length_index);
    int pos_index = 0;
    mask.push_back(0);
    mask.push_back(length_helper);
    bool again = false;
    for(int i=1;i<input.length();i++){
        if(letter != input[i]){
            length_helper = length_helper - apperance;
            while(length_helper < 0){
                if(again){
                    mask.push_back(length.at(length_index) +100);
                }
                length_index++;
                mask.push_back(outcome.length());
                mask.push_back(length.at(length_index));
                length_helper = length_helper + length.at(length_index);
                again = true;
            }
            again = false;
            outcome = outcome + letter + std::to_string(apperance);
            letter = input[i];
            apperance = 1;
            if(pos.at(pos_index)==i){
                    mask.push_back(apperance +100);
                    pos_index++;
            }
        }
        else{
            apperance++;
            //std::cout<< i<< std::endl;
            //std::cout<< pos.at(pos_index)<< std::endl;
            if(pos.at(pos_index)==i){
                if(length.at(pos_index) > apperance){
                    mask.push_back(apperance+100);
                }
                pos_index++;
            }
        }
    }
            std::cout<< length_helper<<std::endl;
            std::cout<< apperance<<std::endl;
            length_helper = length_helper - apperance;
            while(length_helper < 0){
                if(again){
                    mask.push_back(length.at(length_index) +100);
                }
                length_index++;
                mask.push_back(outcome.length());
                mask.push_back(length.at(length_index));
                length_helper = length_helper + length.at(length_index);
                if(length_helper == 0){
                    mask.push_back(length.at(length_index) +100);
                }
                again = true;
            }
            again = false;
            outcome = outcome + letter + std::to_string(apperance);
}




void generate2(){
    std::cout << "Werte: 7776, 66, 644, 4, 445, 648, 8822, 2"<< std::endl;
    std::cout << "input: 77766664444564888222"<< std::endl;
    std::cout << "output: 7364445161418323" << std::endl;
    std::cout << "Maske: 0,4,2,2,2,3,4,1,4,2,8,3,10,4,12,1"<< std::endl;
}


void generate(std::vector<int> &start){
    for(int i=0;i<10;i++){
        start.push_back(rand() % 1024);
    }
}


int main()
{
     std::vector<int> start;
     std::vector<int> mask;
    std::string outcome = "";
    
    generate(start);
    for (auto i: start)
        std::cout << i << ", ";
    std::cout<<std::endl;
    
    encode(start, mask, outcome);
    
    std::cout << outcome << std::endl;
    for (auto i: mask)
        std::cout << i << ", ";
 
    
    hello_world<<<1, 1>>>();
    hipDeviceReset();
    
    
    
    
    
}