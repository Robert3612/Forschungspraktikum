
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
template <typename T>
using EnableIfIntegral = std::enable_if_t<std::is_integral<T>::value>;

template <size_t T>
class Slab
{
private:
    std::bitset<T> bitset;
    std::vector<int> valueBits;

public:

    std::vector<int> getValueBits(){
        return valueBits;
    }
    
    std::bitset<T> getBitset() {
        return bitset;
    }
    
    template <size_t E>
    static std::vector<Slab<E>> packBitsets(const std::vector<std::string> &values)
    {

        auto is_too_big = [](std::string i)
        { return i.length() > T; };

        if (auto h = std::find_if(values.begin(), values.end(), is_too_big); h != values.end())
        {
            std::cout << "Error: Integers are bigger than bitset" << std::endl;
            exit(EXIT_FAILURE);
        }

        int bit_position = 0;
        auto slabs = std::vector<Slab<E>>();

        auto slab = Slab<E>();

        for (auto s : values)
        {
            if ((bit_position + s.length()) > T)
            {
                slabs.push_back(slab);
                slab = Slab<E>();
                bit_position = 0;
            }

            for (auto &ch : s)
            {
                slab.bitset.set(bit_position, ch == '1');
                bit_position++;
            }

            slab.valueBits.push_back(s.length());
        }

        slabs.push_back(slab);

        return slabs;
    }

    std::vector<uint64_t> unpack()
    {

        auto output = std::vector<uint64_t>();

        int start_bit = 0;

        for (auto end_bit : valueBits)
        {
            std::string s;

            for (auto i = start_bit; i < end_bit + start_bit; i++)
            {
                auto h = bitset[i];
                if (h)
                    s.push_back('1');
                else
                    s.push_back('0');
            }

            output.push_back(std::stoi(s, nullptr, 2));
            start_bit = end_bit;
        }

        return output;
    }
};

class GPU_Slab
{
private:
    bool* bitset;
    int bitset_length;
    int* valueBits;
    int valueBits_length;
public:

    GPU_Slab(std::string CPU_bitset, std::vector<int> CPU_valueBits){
        bitset = (bool*)malloc(CPU_bitset.size() * sizeof(bool));
        for(int i=0;i<CPU_bitset.size();i++){
            if(CPU_bitset[i]=='1'){
                bitset[i] = true;
            }
            else{
                bitset[i] = false;
            }
        }
        bitset_length = CPU_bitset.size();

        valueBits = (int*)malloc(CPU_valueBits.size() * sizeof(int));
        for(int i=0;i<CPU_valueBits.size();i++){
            valueBits[i] = CPU_valueBits[i];
        }
        valueBits_length = CPU_valueBits.size();
    }

    __host__ __device__ int* getValueBits(){
        return valueBits;
    }
    
    __host__ __device__ bool* getBitset() {
        return bitset;
    }

    __host__ __device__ int getValueBits_length(){
        return valueBits_length;
    }
    
    __host__ __device__ int getBitset_length() {
        return bitset_length;
    }
};

__global__ 
void add(GPU_Slab *slabs, int elementcount){

    for(int i=0;i<elementcount;i++){

        printf( "%d,", (slabs[i].getValueBits_length()));
    for (int j=0;j< slabs[i].getValueBits_length();j++){
        printf( "%d,", (slabs[i].getValueBits())[j]);
    }

    /**  
    auto bitset = slab.getBitset();
    for(int i=0;i<32;i++)
        printf( "%d,", bitset[i])
    printf( "\n")**/
    }
    
    
}



void removeLeadingZeros(std::vector<std::string> &vector)
{

    for (auto i = 0; i < vector.size(); i++)
    {
        auto pos = vector[i].find('1');
        if (pos > 0)
            vector[i].erase(0, pos);
    }
}

template <size_t E>
GPU_Slab* change(std::vector<Slab<E>> CPU_Slab){
    GPU_Slab* r = (GPU_Slab*)malloc( CPU_Slab.size()* sizeof(GPU_Slab));
    for(int i=0;i<CPU_Slab.size();i++){
        r[i] = GPU_Slab(CPU_Slab[i].getBitset().to_string(), CPU_Slab[i].getValueBits());
    }
    return r;
}

int main()
{
    // 2465,417
    std::vector<std::string> h{"100110100001", "000110100001","100000000000001","000100000001010101010101", "10101", "1010"};

    removeLeadingZeros(h);

    auto slabs = Slab<32>::packBitsets<32>(h);
    std::cout<< slabs.size() << std::endl;

    GPU_Slab* gpu_slaps = change(slabs);

    for(int i=0;i<slabs.size();i++){
        for (int j=0;j< gpu_slaps[i].getValueBits_length();j++){
            printf( "%d,", (gpu_slaps[i].getValueBits())[j]);
        }
        printf( "\n");
    }
    printf( "\n");

    GPU_Slab* d_slab;
    hipMalloc(&d_slab, slabs.size() * sizeof(GPU_Slab));
    hipMemcpy(d_slab, gpu_slaps, slabs.size() * sizeof(GPU_Slab), hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_slab, slabs.size());
    hipDeviceReset();

    //auto slab = slabs.front();
    /**
    for(auto slab: slabs){
    for (auto s : slab.getValueBits())
        std::cout << s << ",";
    
    std::cout<<std::endl;
        
    auto bitset = slab.getBitset();
    for(int i=0;i<32;i++)
        std::cout << bitset[i] << ",";
    std::cout<<std::endl;
    }**/
}