
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>


class Slab
{
private:
    std::bitset<64> bitset;


public:
    static Slab packBitsets(const std::vector<std::string> &values, int n)
    {
        if(64 % n != 0){
            std::cout << "Error: Integers are bigger than bitset" << std::endl;
            exit(EXIT_FAILURE);
        }

        Slab slab = Slab();
        int pos = 0;
        for(auto value: values){
            for( auto ch: value){
                slab.bitset.set(pos, ch == '1');
                pos++;
            }
        }
        return slab;
    }

    std::vector<uint64_t> unpack(int n)
    {

        auto output = std::vector<uint64_t>();

        int start_bit = 0;
        int amount = 64 %n;

        for (int j =0;j<amount ;j++)
        {
            std::string s;
            start_bit = j;

            for (auto i = start_bit; i < n + start_bit; i++)
            {
                auto h = bitset[i];
                if (h)
                    s.push_back('1');
                else
                    s.push_back('0');
            }

            output.push_back(std::stoi(s, nullptr, 2));
        }

        return output;
    }
};



void removeLeadingZeros(std::vector<std::string> &vector, int max)
{
    int length;
    for (auto i = 0; i < vector.size(); i++)
    {
        length = vector[i].size() - max;
        auto pos = vector[i].find('1');
        if (pos > 0)
            vector[i].erase(0, pos);
    }
}

int main()
{
    int bitLength = 8;
    // 2465,417
    std::vector<std::string> h{"10100001", "10100001", "110101010", "11110000"};

    //removeLeadingZeros(h, 11);

    auto slab = Slab::packBitsets(h, 8);

    for(auto s: slab.unpack(8))
    std::cout<< s<<std::endl;
}