
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>
__device__ void get_number_and_length(char *array, int* mask, int &number_length, int &number, int &helper, int i, int h){
    //printf("hallo37 %u %u %u %u %u\n", array[12], array[13], helper, i, h);
    number = array[helper] - '0';
    //printf("hallo36 %u %u %u %u\n", number_length, number, helper, i);
    int otherHelper = helper + 1;
    int numbercount = 0;
    number_length  = 0;
    int n = 0;
    while(array[otherHelper] != 'E'){
        otherHelper++;
        numbercount++;
    }
    otherHelper = helper + 1;
    helper = helper + 2 + numbercount;
    while(numbercount > 0){
        n =array[otherHelper] - '0';
        number_length = number_length + n *pow(10, numbercount-1);
        otherHelper++;
        numbercount--;
    }
    //printf("hallo38 %u %u %u %u %u\n", number_length, number, helper, i, h);
}

   __device__ uint64_t decode_int(char *array, int* mask, int i, int h)
{
    //printf("hallo9 %u %u %u\n", i, h, 9);
    int helper;
    int length;
    int number_length = 0;
    uint64_t answer = 0;
    //int otherHelper = helper +1;
    int number = 0;

    helper = mask[i*3]; //0
    get_number_and_length(array, mask, number_length, number, helper, i, h);
    //printf("hallo39 %u %u %u %u\n", number_length, number, helper, i);
    //int number = array[helper]; //1
    //int numbercount = 0;
    //while(array[otherHelper] != 'E'){
      //  number_length = number_length + array[otherHelper] *pow(10, numbercount);
      //  otherHelper++;
        //numbercount++;
    //}
    //helper = helper + 2 + numbercount;
    //number_length = array[helper+1]; //1
    length = mask[i*3+1]; //3
    //printf("hallo10");
    if(length == mask[i*3+2]){// 3==1 
        for(int j=0;j< length;j++){
            answer = answer +(uint64_t) number * pow(10,j);
        }
        return answer;
    }else{
        for(int j = i-1;j>=0;j--){
            if(mask[j*3+2] == 0){
                break;
            }
        number_length = number_length - mask[j*3+2];
        if(mask[j*3+1] - mask[j*3+2] != 0){
            break;
        }
    }
    //printf("hallo39 %u %u %u %u %u\n", number_length, number, helper, i, h);
    //printf("hallo11 \n");
        int k=length;
        while(k>0){ //3
        //printf("hallo41 %u %u %u %u\n", number_length, number, k, i);
        //printf("hallo12 %u %u %u\n", k, length, 9);
            for(int j=0;j<number_length;j++){
                //printf("hallo13 %u %u %u\n", j, number_length, 9);
                answer = answer +(uint64_t) number * pow(10, k-1);
                k--;
            }
            if(k > 0){
            //printf("hallo9 %u %u %u\n", i, h, helper);
            get_number_and_length(array, mask, number_length, number, helper, i,h);
            //printf("hallo40 %u %u %u %u %u\n", number_length, number, helper,k, i);
            //number = array[helper];
            //number_length = array[helper+1];
            }
        }
    }

    return answer;
}



__global__ 
void add(char *A, char *B, int *C, int *mask_A, int *mask_B, int elementcount) {
    //printf("hallo \n");
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {   
        //printf("hallo5 %u %u %u\n", A[60], B[75], 2);
        a = decode_int(A, mask_A, i, 0);
        b = decode_int(B, mask_B, i, 1);

        printf("hallo4 %u %lld %lld\n", i, a, b);
        if(a > b){
            C[i] = (int) 1;
        }
        else{
            C[i] = (int) 0;
        }
    }
    
}


__global__
void hello_world(char *A, int *mask_A, int i)
{int a = decode_int(A, mask_A, i, 0);
 printf("Hello World From GPU!\n");
 printf ("answer: %d \n", a);
}

std::vector<uint64_t> compare_cpu(std::vector<uint64_t> a, std::vector<uint64_t> b){
    std::vector<uint64_t> C;
    for(int i=0;i<a.size();i++){
        std::cout<<i<<", " << a.at(i)<<", "<<b.at(i)<<std::endl;
        if(a.at(i) > b.at(i)){
            C.push_back(1);
        }
        else{
            C.push_back(0);
        }
    }
    return C;
}

void encode2(std::vector<uint64_t> start, std::vector<int> &mask, std::string &outcome){
    //std::string input = std::to_string(start.at(0));
    std::string input = "";
    std::string input2 = "";
    for(int i=0;i<start.size();i++){
        input = input + std::to_string(start.at(i));
        input2 = input2 + std::to_string(start.at(i)) + 'A';
    }
    char helper;
    int count = 0;
    int step = 0;
    int max_count = 0;
    int test_count=0;
    mask.push_back(0);
    for(int i = 0;i<input.length();i++){
        if(input2[step]== 'A'){
            mask.push_back(max_count);
            if(input[i] != input[i-1]){
                mask.push_back(0);
            }
            else{
                mask.push_back(test_count);
            }
        }
        if(count == 0){
            helper = input[i];
            count = 1;
        }
        else{
            if(helper == input[i]){
                count++;
            }
            else{
                //if(count > 9){
                //    outcome = outcome + helper + 'A' + std::to_string(count);
                //}
                //else{
                    outcome = outcome + helper + std::to_string(count) + "E";
                //}
                helper = input[i];
                count = 1;
                test_count =0;
            }
        }
        if(input2[step]== 'A'){
            mask.push_back(outcome.length());
            max_count = 0;
            step++;
            test_count=0;
        }
        max_count++;
        step++;
        test_count++;
    }
    mask.push_back(max_count);
    mask.push_back(0);

        //if(count > 9){
          //          outcome = outcome + helper + 'B' + std::to_string(count);
            //    }
              //  else{
                    outcome = outcome + helper + std::to_string(count) + "E";
                //}
    
}


void generate2(){
    std::cout << "Werte: 7776, 66, 644, 4, 445, 648, 8822, 2"<< std::endl;
    std::cout << "input: 77766664444564888222"<< std::endl;
    std::cout << "output: 7364455161418323" << std::endl;
    std::cout << "Maske: 0,4,2,2,2,3,4,1,4,2,8,3,10,4,12,1"<< std::endl;
}


void generate(std::vector<uint64_t> &start,int elementcount){
    /**
    for(int i=0;i<elementcount;i++){
        start.push_back(rand() % 1024);
    }
    **/
    
    start.push_back(988888888888888);
    start.push_back(866);
    start.push_back(666);
    start.push_back(6);
    start.push_back(666);
    start.push_back(666);
    start.push_back(8822);
    start.push_back(2);
    

}

void validate(std::vector<uint64_t> h, int* d) {
    for (size_t i = 0; i < h.size(); i++) {
        if (h.at(i) != d[i]) {
            std::cout << "found invalidated field in element " << i << std::endl;
            std::cout << "on CPU side: " << h.at(i) << std::endl;
            std::cout << "on GPU side: " << d[i] << std::endl;
            
            
        }
    }
}

char* to_char_array(std::string s){
    char* answer = (char*)malloc(s.length() * sizeof(char));
    for(int i=0;i<s.length();i++){
        answer[i] = s[i];
    }
    return answer;
}

int* to_int_array(std::vector<int> v){
    int* answer = (int*)malloc(v.size() * sizeof(int));
    for(int i=0;i<v.size();i++){
        answer[i] = v.at(i);
    }
    return answer;
}


int main()
{
     std::vector<uint64_t> start;
     std::vector<int> mask;
    std::string outcome = "";

    std::vector<uint64_t> start2;
    std::vector<int> mask2;
    std::string outcome2 = "";

    int elementcount = 8;
    
    generate(start, elementcount);
    generate(start2, elementcount);
    //for (auto i: start)
    //    std::cout << i << ", ";
    //std::cout<<std::endl;

    //for (auto i: start2)
    //    std::cout << i << ", ";
    //std::cout<<std::endl;
    encode2(start, mask, outcome);
    encode2(start2, mask2, outcome2);
    
    //std::cout << outcome << std::endl;
    /**
    for (auto i: mask)
        std::cout << i << ", ";
    std::cout<<std::endl;

    std::cout << outcome2 << std::endl;

    for (auto i: mask2)
        std::cout << i << ", ";
    std::cout<<std::endl;
    **/
    
    char* A;
    char* d_A;
    int* mask_A;
    int* d_mask_A;
    char* B;
    char* d_B;
    int* mask_B;
    int* d_mask_B;

    int* d_C;

    int* h_out;
    size_t bytes1 = outcome.length() * sizeof(char);
    size_t bytes3 = outcome2.length() * sizeof(char);
    size_t bytes2 = mask.size() * sizeof(int);
    size_t bytes4 = mask2.size() * sizeof(int);

    h_out = (int*)malloc(elementcount*sizeof(int));

    A = (char*)malloc(bytes1);
    mask_A = (int*)malloc(bytes2);

    B = (char*)malloc(bytes3);
    mask_B = (int*)malloc(bytes4);

    hipMalloc(&d_A, bytes1);
    hipMalloc(&d_mask_A, bytes2);

    hipMalloc(&d_B, bytes3);
    hipMalloc(&d_mask_B, bytes4);

    hipMalloc(&d_C, elementcount* sizeof(int));

    A = to_char_array(outcome);
    std::cout<< "hello " << A[0] << ", " << A[1]<< std::endl;
    B = to_char_array(outcome2);
    std::cout<< "hello2 " << B[86] << ", " << B[87]<< std::endl;


    mask_A = to_int_array(mask);
    mask_B = to_int_array(mask2);


    hipMemcpy( d_A, A, bytes1, hipMemcpyHostToDevice);
    hipMemcpy( d_mask_A, mask_A, bytes2, hipMemcpyHostToDevice);
    hipMemcpy( d_B, B, bytes3, hipMemcpyHostToDevice);
    hipMemcpy( d_mask_B, mask_B, bytes4, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, elementcount * sizeof(int));

    add<<<64, 1024>>>(d_A,d_B,d_C, d_mask_A,d_mask_B , elementcount);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    hipMemcpy(h_out, d_C, elementcount* sizeof(int), hipMemcpyDeviceToHost);
    
    std::vector<uint64_t> cpu = compare_cpu(start, start2);

    validate(cpu, h_out);
    
    
}