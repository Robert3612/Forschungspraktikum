
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>
#include <sstream>
#include <inttypes.h>
#include <fstream>
__device__ uint64_t pow10_fast(int n){
    static uint64_t pow10[20] = {
        1, 10, 100, 1000, 10000, 
        100000, 1000000, 10000000, 100000000, 
        1000000000,
        10000000000,
        100000000000,
        1000000000000,
        10000000000000,
        100000000000000,
        1000000000000000,
        10000000000000000,
        100000000000000000,
        1000000000000000000,
        10000000000000000000
    };

    return pow10[n];
}

__device__ void get_number_and_length(char *array, int* mask, int &number_length, int &number, int &helper, int i, int h){
    //printf("hallo37 %u %u %u %u %u\n", array[12], array[13], helper, i, h);
    number = array[helper] - '0';
    //printf("hallo36 %u %u %u %u\n", number_length, number, helper, i);
    int otherHelper = helper + 1;
    int numbercount = 0;
    number_length  = 0;
    int n = 0;
    while(array[otherHelper] != 'E'){
        otherHelper++;
        numbercount++;
    }
    otherHelper = helper + 1;
    helper = helper + 2 + numbercount;
    while(numbercount > 0){
        n =array[otherHelper] - '0';
        number_length = number_length + n *pow10_fast( numbercount-1);
        otherHelper++;
        numbercount--;
    }
    //printf("hallo38 %u %u %u %u %u\n", number_length, number, helper, i, h);
}
   __device__ uint64_t decode_int_no(char *array, int* mask, int i, int h)
{

    int helper = mask[i*3];
    int length = mask[i*3+1];
    uint64_t answer = 0;
    int n = 0;
    
    for(int j=length; j>0;j--){
        n = array[helper] - '0';
        answer = answer + (uint64_t) n * pow10_fast(j-1);
        helper++;
    }
    

    return answer;
}


   __device__ uint64_t decode_int(char *array, int* mask, int i, int h)
{
    //printf("hallo9 %u %u %u\n", i, h, 9);
    int helper;
    int length;
    int number_length = 0;
    uint64_t answer = 0;
    //int otherHelper = helper +1;
    int number = 0;

    helper = mask[i*3]; //0
    get_number_and_length(array, mask, number_length, number, helper, i, h);
    //printf("hallo39 %u %u %u %u\n", number_length, number, helper, i);
    //int number = array[helper]; //1
    //int numbercount = 0;
    //while(array[otherHelper] != 'E'){
      //  number_length = number_length + array[otherHelper] *pow(10, numbercount);
      //  otherHelper++;
        //numbercount++;
    //}
    //helper = helper + 2 + numbercount;
    //number_length = array[helper+1]; //1
    length = mask[i*3+1]; //3
    //printf("hallo10");
    if(length == mask[i*3+2]){// 3==1 
        for(int j=0;j< length;j++){
            answer = answer +(uint64_t) number * pow10_fast(j);
        }
        return answer;
    }else{
        for(int j = i-1;j>=0;j--){
            if(mask[j*3+2] == 0){
                break;
            }
        number_length = number_length - mask[j*3+2];
        if(mask[j*3+1] - mask[j*3+2] != 0){
            break;
        }
    }
    //printf("hallo39 %u %u %u %u %u\n", number_length, number, helper, i, h);
    //printf("hallo11 \n");
        int k=length;
        while(k>0){ //3
        //printf("hallo41 %u %u %u %u\n", number_length, number, k, i);
        //printf("hallo12 %u %u %u\n", k, length, 9);
            for(int j=0;j<number_length;j++){
                //printf("hallo13 %u %u %u\n", j, number_length, 9);
                answer = answer +(uint64_t) number * pow10_fast(k-1);
                k--;
                if(k<=0){
                    break;
                }
            }
            if(k > 0){
            //printf("hallo9 %u %u %u\n", i, h, helper);
            get_number_and_length(array, mask, number_length, number, helper, i,h);
            //printf("hallo40 %u %u %u %u %u\n", number_length, number, helper,k, i);
            //number = array[helper];
            //number_length = array[helper+1];
            }
        }
    }

    return answer;
}

__global__ 
void compare_no(char *A, char *B, int *C, int *mask_A, int *mask_B, int elementcount) {
    //printf("hallo \n");
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {   
        //printf("hallo5 %u %u %u\n", A[60], B[75], 2);
        a = decode_int_no(A, mask_A, i, 0);
        b = decode_int_no(B, mask_B, i, 1);

        //printf("hallo4 %u %lld %lld\n", i, a, b);
        if(a > b){
            C[i] = (int) 1;
        }
        else{
            C[i] = (int) 0;
        }
    }
    
}

__global__ 
void add(char *A, char *B, int *C, int *mask_A, int *mask_B, int elementcount) {
    //printf("hallo \n");
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {   
        //printf("hallo5 %u %u %u\n", A[60], B[75], 2);
        a = decode_int(A, mask_A, i, 0);
        b = decode_int(B, mask_B, i, 1);

        //printf("hallo4 %u %lld %lld\n", i, a, b);
        if(a > b){
            C[i] = (int) 1;
        }
        else{
            C[i] = (int) 0;
        }
    }
    
}


__global__
void hello_world(char *A, int *mask_A, int i)
{int a = decode_int(A, mask_A, i, 0);
 printf("Hello World From GPU!\n");
 printf ("answer: %d \n", a);
}

std::vector<uint64_t> compare_cpu(std::vector<uint64_t> a, std::vector<uint64_t> b){
    std::vector<uint64_t> C;
    for(int i=0;i<a.size();i++){
        //std::cout<<i<<", " << a.at(i)<<", "<<b.at(i)<<std::endl;
        if(a.at(i) > b.at(i)){
            C.push_back(1);
        }
        else{
            C.push_back(0);
        }
    }
    return C;
}
void encode_no(std::vector<uint64_t> start, std::vector<int> &mask, std::string &outcome){
    std::string helper = "";
    for(int i=0;i<start.size();i++){
        mask.push_back(outcome.length());
        helper = std::to_string(start.at(i));
        mask.push_back(helper.length());
        mask.push_back(0);
        outcome = outcome + helper;
    }
}

void encode2(std::vector<uint64_t> start, std::vector<int> &mask, std::string &outcome){
    //std::string input = std::to_string(start.at(0));
    std::string input = "";
    std::string input2 = "";
    for(int i=0;i<start.size();i++){
        input = input + std::to_string(start.at(i));
        input2 = input2 + std::to_string(start.at(i)) + 'A';
    }
    char helper;
    int count = 0;
    int step = 0;
    int max_count = 0;
    int test_count=0;
    mask.push_back(0);
    for(int i = 0;i<input.length();i++){
        if(input2[step]== 'A'){
            mask.push_back(max_count);
            if(input[i] != input[i-1]){
                mask.push_back(0);
            }
            else{
                mask.push_back(test_count);
            }
        }
        if(count == 0){
            helper = input[i];
            count = 1;
        }
        else{
            if(helper == input[i]){
                count++;
            }
            else{
                //if(count > 9){
                //    outcome = outcome + helper + 'A' + std::to_string(count);
                //}
                //else{
                    outcome = outcome + helper + std::to_string(count) + "E";
                //}
                helper = input[i];
                count = 1;
                test_count =0;
            }
        }
        if(input2[step]== 'A'){
            mask.push_back(outcome.length());
            max_count = 0;
            step++;
            test_count=0;
        }
        max_count++;
        step++;
        test_count++;
    }
    mask.push_back(max_count);
    mask.push_back(0);

        //if(count > 9){
          //          outcome = outcome + helper + 'B' + std::to_string(count);
            //    }
              //  else{
                    outcome = outcome + helper + std::to_string(count) + "E";
                //}
    
}


void generate2(){
    std::cout << "Werte: 7776, 66, 644, 4, 445, 648, 8822, 2"<< std::endl;
    std::cout << "input: 77766664444564888222"<< std::endl;
    std::cout << "output: 7364455161418323" << std::endl;
    std::cout << "Maske: 0,4,2,2,2,3,4,1,4,2,8,3,10,4,12,1"<< std::endl;
}

uint64_t string_to_uint64(std::string str) {
  std::stringstream stream(str);
  uint64_t result;
  stream >> result;
  return result;
}

void generate_stuff(std::vector<uint64_t> &start,int elementcount, int repeat){
    char number ='0'+ rand() % 10 ;
    int single = 0;
    int length = 0;
    std::string realnumber = "";
    int diff = repeat;
    int repeat_diff= repeat;
    for(int i=0;i<elementcount;i++){
        length = rand() % 19 + 1 ;
        single = rand() % 2;
        while(length > 0){
            if(single == 0){
                realnumber = realnumber + std::to_string(rand() % 10);
                length--;
                single = rand() %2;
            }
            else{
                diff = length - repeat_diff;
                if(diff >= 0){
                    realnumber = realnumber + std::string(repeat_diff,number );
                    length = length - repeat_diff;
                    repeat_diff = 0;
                }
                else{
                    realnumber = realnumber + std::string(length,number );
                    repeat_diff = repeat_diff - length;
                    length = 0;
                }
                if(repeat_diff <= 0){
                    repeat_diff = repeat;
                    number ='0'+ rand() % 10 ;
                }
            }
        }
        //std::cout<<realnumber<<std::endl;
        start.push_back(std::stoull(realnumber));
        realnumber = "";
    }
}

void generate_stuff2(std::vector<uint64_t> &start,int elementcount, int repeat, int length){
    char number ='0'+ rand() % 10 ;
    int single = 0;
    std::string realnumber = "";
    int reallength = length;
    while(length > 0){
        //std::cout<<length<<std::endl;
        single = rand() % 2;
        if(single == 0){
            realnumber = realnumber + std::to_string(rand() % 9 +1);
            length--;
        }
        else{
            realnumber = realnumber + std::string(repeat,number );
            length = length - repeat;
            number ='0'+ rand() % 10 ;
        }
    }
    realnumber = realnumber.substr(0,reallength);
    //std::cout << length<<std::endl;
    //std::cout << realnumber.length()<<std::endl;
    int n = realnumber.length() / elementcount;
    std::cout<<realnumber<<std::endl;
    for(int i =0;i<elementcount;i++){
        std::cout<<i<<std::endl;
        std::cout<<n<<std::endl;
        start.push_back(std::stoull(realnumber.substr(i*n,n)));
    }
}

void generate(std::vector<uint64_t> &start,int elementcount){
    
    for(int i=0;i<elementcount;i++){
        start.push_back(rand() % 1024);
    }
    
    /**
    start.push_back(988888888888888);
    start.push_back(866);
    start.push_back(666);
    start.push_back(6);
    start.push_back(666);
    start.push_back(666);
    start.push_back(8822);
    start.push_back(2);
    **/

}

void validate(std::vector<uint64_t> h, int* d) {
    for (size_t i = 0; i < h.size(); i++) {
        if (h.at(i) != d[i]) {
            std::cout << "found invalidated field in element " << i << std::endl;
            std::cout << "on CPU side: " << h.at(i) << std::endl;
            std::cout << "on GPU side: " << d[i] << std::endl;
            
            
        }
    }
}

char* to_char_array(std::string s){
    char* answer = (char*)malloc(s.length() * sizeof(char));
    for(int i=0;i<s.length();i++){
        answer[i] = s[i];
    }
    return answer;
}

int* to_int_array(std::vector<int> v){
    int* answer = (int*)malloc(v.size() * sizeof(int));
    for(int i=0;i<v.size();i++){
        answer[i] = v.at(i);
    }
    return answer;
}

void test(){

    int length = 1000000;
    int elementcount = 500000;
    
    std::ofstream myFile("run-length4.csv");
    myFile << "kernel;element_count;repeat;block_count;thread_count;time_ms;throughput;length\n";

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int l = 1; l<=50;l++){
        std::cout<< "l: "<<l<<std::endl;

        std::vector<uint64_t> start_run;
        std::vector<int> mask;
        std::string outcome = "";

        std::vector<uint64_t> start_run2;
        std::vector<int> mask2;
        std::string outcome2 = "";

        generate_stuff2(start_run, elementcount, l, length);
        generate_stuff2(start_run2, elementcount, l, length);

        encode2(start_run, mask, outcome);
        encode2(start_run2, mask2, outcome2);

    char* A;
    char* d_A;
    int* mask_A;
    int* d_mask_A;
    char* B;
    char* d_B;
    int* mask_B;
    int* d_mask_B;

    int* d_C;

    int* h_out;
    size_t bytes1 = outcome.length() * sizeof(char);
    size_t bytes3 = outcome2.length() * sizeof(char);
    size_t bytes2 = mask.size() * sizeof(int);
    size_t bytes4 = mask2.size() * sizeof(int);

    h_out = (int*)malloc(elementcount*sizeof(int));

    A = (char*)malloc(bytes1);
    mask_A = (int*)malloc(bytes2);

    B = (char*)malloc(bytes3);
    mask_B = (int*)malloc(bytes4);

    hipMalloc(&d_A, bytes1);
    hipMalloc(&d_mask_A, bytes2);

    hipMalloc(&d_B, bytes3);
    hipMalloc(&d_mask_B, bytes4);

    hipMalloc(&d_C, elementcount* sizeof(int));

    A = to_char_array(outcome);
    B = to_char_array(outcome2);


    mask_A = to_int_array(mask);
    mask_B = to_int_array(mask2);


    hipMemcpy( d_A, A, bytes1, hipMemcpyHostToDevice);
    hipMemcpy( d_mask_A, mask_A, bytes2, hipMemcpyHostToDevice);
    hipMemcpy( d_B, B, bytes3, hipMemcpyHostToDevice);
    hipMemcpy( d_mask_B, mask_B, bytes4, hipMemcpyHostToDevice);

    for(int i=16;i<=1024;i=i*2){
        std::cout<< "i: "<<i<<std::endl;
        for(int j=8;j<=512;j=j*2){
            std::cout<<"j: "<< j<<std::endl;
    hipMemset(d_C, 0, elementcount * sizeof(int));

    hipEventRecord(start);
    add<<<j, i>>>(d_A,d_B,d_C, d_mask_A,d_mask_B , elementcount);
    hipEventRecord(stop);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    hipMemcpy(h_out, d_C, elementcount* sizeof(int), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    std::vector<uint64_t> cpu = compare_cpu(start_run, start_run2);

    validate(cpu, h_out);

    uint64_t input_size = (outcome.length() + mask.size() * sizeof(int) ) + (outcome2.length() + mask2.size() * sizeof(int) );
    myFile << "no-run-length4" << ";";
    myFile << elementcount << ";";
    myFile << l << ";";
    myFile << j << ";";
    myFile << i << ";";
    myFile << milliseconds << ";";
    myFile << input_size/milliseconds/1e6 << ";";
    myFile << outcome.length() + outcome2.length() <<"\n";
      }
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_mask_A);
    hipFree(d_mask_B);

    free(h_out);
    }
}


int main()
{   
    test();
    /**
     std::vector<uint64_t> start;
     std::vector<int> mask;
    std::string outcome = "";

    std::vector<uint64_t> start2;
    std::vector<int> mask2;
    std::string outcome2 = "";

    int elementcount = 10;
    
    generate_stuff2(start, elementcount, 5, 20);
    generate_stuff2(start2, elementcount, 5, 20);
    for (auto i: start)
        std::cout << i << ", ";
    std::cout<<std::endl;

    for (auto i: start2)
        std::cout << i << ", ";
    std::cout<<std::endl;
    /**
    encode2(start, mask, outcome);
    encode2(start2, mask2, outcome2);
    
    std::cout << outcome << std::endl;
    
    for (auto i: mask)
        std::cout << i << ", ";
    std::cout<<std::endl;

    std::cout << outcome2 << std::endl;

    for (auto i: mask2)
        std::cout << i << ", ";
    std::cout<<std::endl;
    
    
    char* A;
    char* d_A;
    int* mask_A;
    int* d_mask_A;
    char* B;
    char* d_B;
    int* mask_B;
    int* d_mask_B;

    int* d_C;

    int* h_out;
    size_t bytes1 = outcome.length() * sizeof(char);
    size_t bytes3 = outcome2.length() * sizeof(char);
    size_t bytes2 = mask.size() * sizeof(int);
    size_t bytes4 = mask2.size() * sizeof(int);

    h_out = (int*)malloc(elementcount*sizeof(int));

    A = (char*)malloc(bytes1);
    mask_A = (int*)malloc(bytes2);

    B = (char*)malloc(bytes3);
    mask_B = (int*)malloc(bytes4);

    cudaMalloc(&d_A, bytes1);
    cudaMalloc(&d_mask_A, bytes2);

    cudaMalloc(&d_B, bytes3);
    cudaMalloc(&d_mask_B, bytes4);

    cudaMalloc(&d_C, elementcount* sizeof(int));

    A = to_char_array(outcome);
    std::cout<< "hello " << A[0] << ", " << A[1]<< std::endl;
    B = to_char_array(outcome2);
    std::cout<< "hello2 " << B[86] << ", " << B[87]<< std::endl;


    mask_A = to_int_array(mask);
    mask_B = to_int_array(mask2);


    cudaMemcpy( d_A, A, bytes1, cudaMemcpyHostToDevice);
    cudaMemcpy( d_mask_A, mask_A, bytes2, cudaMemcpyHostToDevice);
    cudaMemcpy( d_B, B, bytes3, cudaMemcpyHostToDevice);
    cudaMemcpy( d_mask_B, mask_B, bytes4, cudaMemcpyHostToDevice);
    cudaMemset(d_C, 0, elementcount * sizeof(int));

    add<<<64, 1024>>>(d_A,d_B,d_C, d_mask_A,d_mask_B , elementcount);
    cudaError_t cudaerr = cudaDeviceSynchronize();
    if (cudaerr != cudaSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               cudaGetErrorString(cudaerr));

    cudaMemcpy(h_out, d_C, elementcount* sizeof(int), cudaMemcpyDeviceToHost);
    
    std::vector<uint64_t> cpu = compare_cpu(start, start2);

    validate(cpu, h_out);

    cudaFree(d_A);
    cudaFree(d_B);
    cudaFree(d_C);
    cudaFree(d_mask_A);
    cudaFree(d_mask_B);

    free(h_out);
    **/
    
}