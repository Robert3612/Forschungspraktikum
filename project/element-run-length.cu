
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>

   __device__ int decode_int(int *array, int* mask, int i)
{
    int helper;
    int length;
    int number_length;
    int answer = 0;


    helper = mask[i*3];
    int number = array[helper];
    number_length = array[helper+1];
    length = mask[i*3+1];
    
    if(length == mask[i*3+2]){
        for(int j=0;j< length;j++){
            answer = answer + number * pow(10,j);
        }
    }else{
        for(int j = i-1;j>=0;j--){
        number_length = number_length - mask[j*3+2];
        if(mask[j*3+1] - mask[j*3+2] != 0){
            break;
        }
    }
        while(length > 0){
            for(int j=0;j<number_length;j++){
                answer = answer + number * pow(10, length-1);
                length--;
                if(length < 0){
                    break;
                }
            }
            if(length > 0){
            helper = helper +2;
            number = array[helper];
            number_length = array[helper+1];
            }
        }
    }

    return answer;
}




__global__ 
void add(int *A, int *B, int *C, int *mask_A, int *mask_B, int elementcount) {
    
    int a;
    int b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int(A, mask_A, i);
        b = decode_int(B, mask_B, i);
        if(a > b){
            C[i] = 1;
        }
        else{
            C[i] = 0;
        }
    }
    
}


__global__
void hello_world(int *A, int *mask_A, int i)
{int a = decode_int(A, mask_A, i);
 printf("Hello World From GPU!\n");
 printf ("answer: %d \n", a);
}

void encode(std::vector<int> start, std::vector<int> &mask, std::string &outcome){
    std::string input = std::to_string(start.at(0));
    std::vector<int> length;
    length.push_back(std::to_string(start.at(0)).length());
    std::vector<int> pos;
    pos.push_back(std::to_string(start.at(0)).size()-1);
    for(int i=1;i<start.size();i++){
        input = input + std::to_string(start.at(i));
        length.push_back(std::to_string(start.at(i)).length());
        pos.push_back(pos.back() + std::to_string(start.at(i)).length());
    }
    
    char letter = input[0];
    int apperance = 1;
    int length_index = 0;
    int length_helper = length.at(length_index);
    int pos_index = 0;
    mask.push_back(0);
    mask.push_back(length_helper);
    bool again = false;
    for(int i=1;i<input.length();i++){
        if(letter != input[i]){
            length_helper = length_helper - apperance;
            while(length_helper < 0){
                if(again){
                    mask.push_back(length.at(length_index) +100);
                }
                length_index++;
                mask.push_back(outcome.length());
                mask.push_back(length.at(length_index));
                length_helper = length_helper + length.at(length_index);
                again = true;
            }
            again = false;
            outcome = outcome + letter + std::to_string(apperance);
            letter = input[i];
            apperance = 1;
            if(pos.at(pos_index)==i){
                    mask.push_back(apperance +100);
                    pos_index++;
            }
        }
        else{
            apperance++;
            //std::cout<< i<< std::endl;
            //std::cout<< pos.at(pos_index)<< std::endl;
            if(pos.at(pos_index)==i){
                if(length.at(pos_index) > apperance){
                    mask.push_back(apperance+100);
                }
                pos_index++;
            }
        }
    }
            std::cout<< length_helper<<std::endl;
            std::cout<< apperance<<std::endl;
            length_helper = length_helper - apperance;
            while(length_helper < 0){
                if(again){
                    mask.push_back(length.at(length_index) +100);
                }
                length_index++;
                mask.push_back(outcome.length());
                mask.push_back(length.at(length_index));
                length_helper = length_helper + length.at(length_index);
                if(length_helper == 0){
                    mask.push_back(length.at(length_index) +100);
                }
                again = true;
            }
            again = false;
            outcome = outcome + letter + std::to_string(apperance);
}




void generate2(){
    std::cout << "Werte: 7776, 66, 644, 4, 445, 648, 8822, 2"<< std::endl;
    std::cout << "input: 77766664444564888222"<< std::endl;
    std::cout << "output: 7364445161418323" << std::endl;
    std::cout << "Maske: 0,4,2,2,2,3,4,1,4,2,8,3,10,4,12,1"<< std::endl;
}


void generate(std::vector<int> &start){
    for(int i=0;i<10;i++){
        start.push_back(rand() % 1024);
    }
}


int main()
{/**
     std::vector<int> start;
     std::vector<int> mask;
    std::string outcome = "";
    
    generate(start);
    for (auto i: start)
        std::cout << i << ", ";
    std::cout<<std::endl;
    
    encode(start, mask, outcome);
    
    std::cout << outcome << std::endl;
    for (auto i: mask)
        std::cout << i << ", ";
    **/
 
    int* A;
    int* d_A;
    int* mask;
    int* d_mask;
    size_t bytes = 6 * sizeof(int);

    A = (int*)malloc(bytes);
    mask = (int*)malloc(9 * sizeof(int));

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_mask, 9 * sizeof(int));

    A[0] = 7;
    A[1] = 3;
    A[2] = 6;
    A[3] = 4;
    A[4] = 4;
    A[5] = 2;

    mask[0] = 0;
    mask[1] = 4;
    mask[2] = 1;
    mask[3] = 2;
    mask[4] = 2;
    mask[5] = 2;
    mask[6] = 2;
    mask[7] = 3;
    mask[8] = 0;


    hipMemcpy( d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_mask, mask, 9 * sizeof(int), hipMemcpyHostToDevice);

    hello_world<<<1, 1>>>(d_A, d_mask, 0);
    hipDeviceReset();
    
    
    
    
    
}