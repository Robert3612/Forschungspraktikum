
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>

std::string decode(std::string s);
int unElias(std::string code);

template <typename T>
using EnableIfIntegral = std::enable_if_t<std::is_integral<T>::value>;


template <size_t T>
class Slab
{
private:
    std::bitset<T> bitset;
    std::vector<int> valueBits;

public:
    template <size_t E>
    static std::vector<Slab<E>> packBitsets(const std::vector<std::string> &values)
    {

        auto is_too_big = [](std::string i)
        { return i.length() > T; };

        if (auto h = std::find_if(values.begin(), values.end(), is_too_big); h != values.end())
        {
            std::cout << "Error: Integers are bigger than bitset" << std::endl;
            exit(EXIT_FAILURE);
        }

        int bit_position = 0;
        auto slabs = std::vector<Slab<E>>();

        auto slab = Slab<E>();

        for (auto s : values)
        {
            if ((bit_position + s.length()) > T)
            {
                slabs.push_back(slab);
                slab = Slab<E>();
            }

            for (auto &ch : s)
            {
                slab.bitset.set(bit_position, ch == '1');
                bit_position++;
            }

            slab.valueBits.push_back(s.length());
        }

        slabs.push_back(slab);

        return slabs;
    }

    std::vector<uint64_t> unpack()
    {

        auto output = std::vector<uint64_t>();

        int start_bit = 0;
        std::string outputS;
        for (auto end_bit : valueBits)
        {
            std::string s;

            for (auto i = start_bit; i < end_bit + start_bit; i++)
            {
                auto h = bitset[i];
                if (h)
                    s.push_back('1');
                else
                    s.push_back('0');
            }
            outputS = decode(s);
            output.push_back(std::stoi(outputS, nullptr, 2));
            start_bit = end_bit;
        }

        return output;
    }
};

std::string EliasGamma(int apperance)
{
    int x = apperance;
    int y;
    std::string answer = "";

do {
    y = x;
    x = x&(x-1);
}while(x);

int result = log2 (y);

for(int i =0; i< result;i++){
    answer = answer + "0";
}
answer = answer + "1";

int rest = apperance - y;

std::string binary = std::bitset<8>(rest).to_string();


binary.erase(0, binary.length()-result);
answer = answer + binary;

return answer;

}

void runLengthEliasGamma(std::vector<std::string> &vector)
{   
    for (auto i = 0; i < vector.size(); i++)
    {
        
        char letter = vector[i][0];
        int apperance = 1;
        std::string newString = "";
        for(int j=1;j<vector[i].length();j++) {
            if(letter != vector[i][j]){
                newString = newString + letter + EliasGamma(apperance);
                letter = vector[i][j];
                apperance=1;
            }
            else{
                apperance++;
            }
        }
        newString = newString + letter + EliasGamma(apperance);
        vector[i] = newString;
    }
}

int unElias(std::string code){
    if(code != ""){
    int length = code.length();
    int answer = pow(2, length);
    answer = answer + std::stoi(code, nullptr, 2);
    return answer;
    }
    return 1;
}

std::string decode(std::string s){
        char letter;
        bool newL = true;
        int counter =0;
        std::string newString = "";
        for(int j=0;j<s.length();j++) {
            if(newL){
                letter = s[j];
                newL = false;
            }
            else{
                if(s[j] == '1'){
                        int apperance = unElias(s.substr(j+1,counter));
                        for(int c=0;c<apperance;c++){
                            newString = newString+ letter;
                        }
                        j = j+ counter;
                        newL=true;
                        counter=0;
                }
                else{
                    counter++;
                }
        }
        }
        return newString;
}


int main()
{
    // 2465,417
    std::vector<std::string> h{"10011010000101001", "000110100001"};
    for (auto s : h)
        std::cout << s << std::endl;

    runLengthEliasGamma(h);
    
    for (auto s : h)
        std::cout << s << std::endl;


    auto slabs = Slab<64>::packBitsets<64>(h);

    auto slab = slabs.front();

    for (auto s : slab.unpack())
        std::cout << s << std::endl;
    
}