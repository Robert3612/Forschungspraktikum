
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>
#include <inttypes.h>
#include <fstream>

#if __CUDA_ARCH__ < 600
__device__ uint64_t myAtomicAdd(uint64_t* address, uint64_t val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        (unsigned long long int)(val +
                              (uint64_t) (assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return (old);
}
#endif

__device__ 
uint64_t decode_int(uint64_t* array, int i, int number_length)
{
    int amount = 64 / number_length;
    int chunk = i/amount;
    int position = i % amount;
    uint64_t slab = array[chunk];
    uint64_t mask = pow(2,number_length) -1;
    mask = mask << (position*number_length);
    uint64_t answer = slab & mask;
    answer = answer >> (position*number_length);
    return answer;
    
}

__device__ 
int getLength(uint64_t bits ){

    int size = 0;

    for (; bits != 0; bits >>= 1){

    size++;
    }

    return size;
}

__device__
void add( uint64_t* C, uint64_t a, uint64_t b, int number_length, int i){

    uint64_t c = a +b;

    if(getLength(c)<=number_length){

        int amount = 64 / number_length;

        int chunk = i/amount;

        int position = i % amount;

        c = c << (position*number_length);

        myAtomicAdd(&C[chunk], c);
    }

}

__device__
void compare(uint64_t* C, int a, int b, int number_length, int i){
    if(a > b){
        uint64_t helper = 0;
        int amount = 64 / number_length;
        int chunk = i/amount;
        int position = i % amount;
        helper = 1;
        helper = helper << (position*number_length);
        myAtomicAdd(&C[chunk],helper);
    }
}

__global__
void hello_world()
{
 printf("Hello World From GPU!\n");
}

__global__ 
void zero_sup_no(uint64_t* A, uint64_t* B, int number_length_A,int number_length_B, int array_length,int elementcount,  uint64_t* C){
    uint64_t a;
    uint64_t b;
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        
        a = decode_int(A, i, number_length_A);
        b = decode_int(B, i,number_length_B);
       if(i ==6){
        printf("wow, %lld %lld\n", (unsigned long long) a, (unsigned long long) b);
       }

        add(C, a, b, number_length_A, i);



    }
}

__global__ 
void zero_sup_yes(uint64_t* A, uint64_t* B, int number_length, int array_length,int elementcount, uint64_t* C){
    extern __shared__ uint64_t shared_mem[];

   for (int i = threadIdx.x; i < array_length; i += blockDim.x )
    {
        //printf("wow, %u %u %u\n", i, i+ array_length, array_length);
       // printf("wow, %u %u %u\n", i, A[i], B[i]);
         //printf("wow, %lld %lld\n", (unsigned long long) A[i], (unsigned long long) B[i]);
        shared_mem[i] = A[i];
        shared_mem[i+array_length] = B[i];
        if(i==75){
            //printf("wow, %u %lld %u\n", i,(unsigned long long) shared_mem[i], array_length);
        }
        //shared_mem[10000] = 3;
    }
   

    
    __syncthreads();
    
    int a;
    int b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int(shared_mem, i, number_length);
        b = decode_int(shared_mem, i+elementcount,number_length);
        int amount = 64 / number_length;
        int chunk = i/amount;


        add(C, a, b, number_length, i);

    }
}


__global__ 
void zero_sup_yes2(uint64_t* A, uint64_t* B, int number_length, int array_length,int elementcount,  uint64_t* C){
    extern __shared__ uint64_t shared_mem[];
    for (int i = threadIdx.x; i < array_length; i += blockDim.x)
    {
        shared_mem[i] = A[i];
        shared_mem[i+array_length] = B[i];
        shared_mem[i+2*array_length] = 0;

    }
    __syncthreads();
    int a;
    int b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int(shared_mem, i, number_length);
        b = decode_int(shared_mem, i+elementcount,number_length);
        add(shared_mem, a, b, number_length, i+2*elementcount);

    }
    __syncthreads();
    for (int i = threadIdx.x; i < array_length; i += blockDim.x)
    {
        C[i] = shared_mem[i+2*array_length];

    }
}

uint64_t getLength_cpu(uint64_t bits ){
    uint64_t size = 0;

    for (; bits != 0; bits >>= 1)
    size++;

    std::cout<<size<<std::endl;
    return size;
}

std::vector<uint64_t> add_cpu(std::vector<uint64_t> a, std::vector<uint64_t> b, int element_length){
    uint64_t c;
    std::vector<uint64_t> C;
    for(int i=0;i<a.size();i++){
        c = a.at(i) + b.at(i);
        std::cout<< i <<std::endl;
        std::cout<< a.at(i) <<std::endl;
        std::cout<< b.at(i) <<std::endl;
        if(getLength_cpu(c) <= element_length){
            C.push_back(c);
        }
        else{
            C.push_back(0);
        }
    }
    return C;
}

std::vector<uint64_t> compare_cpu(std::vector<uint64_t> a, std::vector<uint64_t> b){
    std::vector<uint64_t> C;
    for(int i=0;i<a.size();i++){
        if(a.at(i) > b.at(i)){
            C.push_back(1);
        }
        else{
            C.push_back(0);
        }
    }
    return C;
}

std::vector<std::string> int_to_string(std::vector<uint64_t> v){
    std::vector<std::string> s;
    for(int i=0;i<v.size();i++){
        s.push_back(std::bitset< 64 >(v.at(i)).to_string());
    }
    return s;
}

std::vector<uint64_t> string_to_int(std::vector<std::string> s){
    uint64_t helper;
    std::vector<uint64_t> v;
    for(int i=0;i<s.size();i++){
        helper = (uint64_t) std::bitset< 64 >(s.at(i)).to_ulong();
        v.push_back(helper);
    }
    return v;
}


struct Slabs{
    int array_length;
    int number_length;
    uint64_t* array;
};


Slabs encode(std::vector<std::string> vector){
    std::vector<uint64_t> helper_array;
    int length = vector.at(0).size();
    int size = 64 / length;
    uint64_t slab = 0;
    uint64_t helper = 0;
    int position = 0;
    Slabs s;
    s.number_length = length;

    for(std::string number:vector){
        helper = (uint64_t) (std::bitset<64>(number)).to_ulong();
        helper = helper << (position*length);
        slab = slab + helper;
        position++;
        if(position == size){
            position = 0;
            helper_array.push_back(slab);
            slab = 0;
        }
    }
    if(slab != 0){
       helper_array.push_back(slab); 
    }

    uint64_t* answer = (uint64_t*)malloc(helper_array.size() * sizeof(uint64_t));
    for(int i=0;i< helper_array.size();i++){
        answer[i] = helper_array.at(i);
    }
    s.array_length = helper_array.size();
    s.array = answer;
    return s;

}

std::vector<uint64_t> decode(uint64_t* numbers, int number_length, int array_length){
    int amount = 64 / number_length;
    uint64_t mask = pow(2,number_length) -1;
    uint64_t slab;
    uint64_t answer;
    std::vector<uint64_t> decoded_numbers;
    for(int i =0;i<array_length;i++){
        slab = numbers[i];
        for(int j =0;j<amount;j++){
            mask = pow(2,number_length) -1;
            mask = mask << (j*number_length);
            answer = slab & mask;
            answer = answer >> (j*number_length);
            decoded_numbers.push_back(answer);
        }
    }
    return decoded_numbers;
}


void removeLeadingZeros(std::vector<std::string> &vector)
{
    int length;
    int max = 0;
    for (auto i = 0; i < vector.size(); i++)
    {
        length = vector[i].size() - vector[i].find('1');
        if(length > max){
            max = length;
        }
    }
    if( max == 0){
        return;
    }
    for(auto i = 0; i < vector.size(); i++){
        if(vector[i].size() > max){
        vector[i].erase(0, vector[i].size() - max);
        }
        else{
            vector[i].insert(0, max - vector[i].size(), '0');
        }
    }
}

void validate(std::vector<uint64_t> h, std::vector<uint64_t> d) {
    for (size_t i = 0; i < h.size(); i++) {
        if (h.at(i) != d.at(i)) {

            std::cout << "found invalidated field in element " << i << std::endl;
            std::cout << "on CPU side: " << h.at(i) << std::endl;
            std::cout << "on GPU side: " << d.at(i) << std::endl;
            
        }
    }
}

void generate(std::vector<uint64_t> &a, std::vector<uint64_t> &b , int n, int elementcount){
    int number = pow(2,n) -1;
    for(int i=0;i<elementcount;i++){
        a.push_back((uint64_t) rand() % number + 1);
        b.push_back((uint64_t) rand() % number + 1);
    }
}

void test(){
    //size_t elementcount=1048576;
    size_t elementcount=5000;
    int length=8;
    std::ofstream myFile("no_shared.csv");
    myFile << "kernel;element_count;bit_count;block_count;thread_count;time_ms;throughput\n";

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int l = 8; l<=64;l=l+8){
        std::cout<< l<<std::endl;
    std::vector<uint64_t> a;
    std::vector<uint64_t> b;

    generate(a, b, l, elementcount);

    std::vector<std::string> h = int_to_string(a);
    std::vector<std::string> h2 = int_to_string(b);

    removeLeadingZeros(h);
    removeLeadingZeros(h2);

    
    Slabs s = encode(h);
    Slabs s2 = encode(h2);


    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;


    uint64_t* h_out;

    size_t bytes = s.array_length * sizeof(uint64_t);
    size_t bytes2 = s.array_length * sizeof(unsigned long long int);
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes2);

    hipMemcpy(d_A, s.array, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, s2.array, bytes, hipMemcpyHostToDevice);
    

    
    for(int i=16;i<=1024;i=i*2){
        std::cout<< i<<std::endl;
        for(int j=8;j<=512;j=j*2){
            std::cout<< j<<std::endl;
            hipMemset(d_C, 0, bytes2);
	
            hipEventRecord(start);
            zero_sup_no<<<64, 1024>>>(d_A, d_B, s.number_length,s2.number_length, s.array_length,h.size(),  d_C);
            hipEventRecord(stop);
    
            hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            int max_number_length;
            int max_array_length;
            if(s.number_length> s2.number_length){
            max_number_length = s.number_length;
            max_array_length = s.array_length;
        }
        else{
            max_number_length = s2.number_length;
            max_array_length = s2.array_length;
        }

        std::vector<uint64_t> decoded_numbers = decode(h_out, max_number_length, max_array_length);
        std::vector<uint64_t> c = add_cpu(a,b, max_number_length);

        validate(c, decoded_numbers);

            myFile << "no_shared" << ";";
            myFile << elementcount << ";";
            myFile << l << ";";
            myFile << j << ";";
            myFile << i << ";";
            myFile << milliseconds << ";";
            myFile << s.array_length*8*2/milliseconds/1e6 << "\n";
        }
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_out);
    }

    
}

int main()
{   
    //test();
    
    std::vector<uint64_t> a;
    std::vector<uint64_t> b;

    generate(a, b,32, 50);

    //std::vector<std::string> h{"00000110000100000000", "110010100000000", "1100000000", "110001100000000", "110101100000001", "110001000000000", "100000100000000", "110101100010000"};
    //std::vector<std::string> h2{"001000100000001", "10010100000001", "10010100000001", "10001100000001", "10101100000000", "10001000000001", "100000100000001", "110101100100001"};
    std::vector<std::string> h = int_to_string(a);
    std::vector<std::string> h2 = int_to_string(b);

    removeLeadingZeros(h);
    removeLeadingZeros(h2);

    for(auto i = 0; i < h.size(); i++){
        //std::cout<< h[i] << std::endl;
        //std::cout<< h2[i] << std::endl;
    }
    //std::vector<uint64_t> a = string_to_int(h);
    //std::vector<uint64_t> b = string_to_int(h2);
    
    Slabs s = encode(h);
    Slabs s2 = encode(h2);
    for(int i=0;i<s.array_length;i++){
        //std::cout<< std::bitset<64>(s.array[i]) << std::endl;
        //std::cout<< std::bitset<64>(s2.array[i]) << std::endl;
    }


    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;


    uint64_t* h_out;

    size_t bytes = s.array_length * sizeof(uint64_t);
    size_t bytes2 = s.array_length * sizeof(unsigned long long int);
    std::cout<< "wow" <<std::endl;
    std::cout<< bytes <<std::endl;
    std::cout<< bytes2 <<std::endl;
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes2);
    for(int i=0;i<s.array_length;i++){
        //std::cout<< i <<", " << s.array[i]<<std::endl;
    }
    hipMemcpy(d_A, s.array, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, s2.array, bytes, hipMemcpyHostToDevice);
    for(int i=0;i<s.array_length;i++){
        //std::cout<< i <<", " << d_B<<std::endl;
    }
    hipMemset(d_C, 0, bytes2);
    std::cout << "lets go" <<std::endl;
    //2*s.array_length*sizeof(uint64_t)
    //64, 1024, 3*s.array_length*sizeof(uint64_t)
    //zero_sup_yes2<<<1, 32, 3*s.array_length*sizeof(uint64_t)>>>(d_A, d_B, s.number_length, s.array_length,h.size(),  d_C);
    zero_sup_no<<<64, 1024>>>(d_A, d_B, s.number_length,s2.number_length, s.array_length,h.size(),  d_C);
    //hello_world<<<1, 1>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));


    hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

    int max_number_length;
    int max_array_length;
    if(s.number_length> s2.number_length){
        max_number_length = s.number_length;
        max_array_length = s.array_length;
    }
    else{
        max_number_length = s2.number_length;
        max_array_length = s2.array_length;
    }
    //for(int i=0;i<max_array_length;i++){
    //    std::cout<< "hello" <<std::endl;
    //    std::cout<< std::bitset<64>(h_out[i]) << std::endl;
    //}
    std::vector<uint64_t> decoded_numbers = decode(h_out, max_number_length, max_array_length);
    //for(int i=0;i<decoded_numbers.size();i++){
    //    std::cout<<decoded_numbers.at(i)<<std::endl;
    //}
    std::vector<uint64_t> c = add_cpu(a,b, max_number_length);

    validate(c, decoded_numbers);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_out);
}