
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>

#if __CUDA_ARCH__ < 600
__device__ uint64_t myAtomicAdd(uint64_t* address, uint64_t val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        (unsigned long long int)(val +
                              (uint64_t) (assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return (old);
}
#endif

__device__ uint64_t decode_int(uint64_t* array, int i, int number_length)
{
    int amount = 64 / number_length;
    int chunk = i/amount;
    int position = i % amount;
    printf("int, %u %u %u %u %u\n", i, number_length, amount, chunk, position);
    uint64_t slab = array[chunk];
    uint64_t mask = pow(2,number_length) -1;
    mask = mask << (position*number_length);
    uint64_t answer = slab & mask;
    answer = answer >> (position*number_length);
    return answer;
    
}

__device__ 
int getLength(int bits ){
    int size = 0;

    for (; bits != 0; bits >>= 1)
    size++;
    printf("i, %d \n", size);
    return size;
}

__device__
void add( uint64_t* C, uint64_t a, uint64_t b, int number_length, int i){
    uint64_t c = a +b;
    if(getLength(c)<=number_length){
        int amount = 64 / number_length;
        int chunk = i/amount;
        int position = i % amount;  
        c = c << (position*number_length);
        myAtomicAdd(&C[chunk], c);
    }
}

__device__
void compare(uint64_t* C, int a, int b, int number_length, int i){
    if(a > b){
        uint64_t helper = 0;
        int amount = 64 / number_length;
        int chunk = i/amount;
        int position = i % amount;
        helper = 1;
        helper = helper << (position*number_length);
        myAtomicAdd(&C[chunk],helper);
    }
}

__global__
void hello_world()
{
 printf("Hello World From GPU!\n");
}

__global__ 
void zero_sup_no(uint64_t* A, uint64_t* B, int number_length, int array_length,int elementcount,  uint64_t* C){
    printf("hallo24324");
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int(A, i, number_length);
        b = decode_int(B, i,number_length);
        printf("a, %d \n", a);
        printf("b, %d \n", b);
        printf("hallo");
        compare(C, a, b, number_length, i);

    }
}

__global__ 
void zero_sup_yes(uint64_t* A, uint64_t* B, int number_length, int array_length,int elementcount, uint64_t* C){
    extern __shared__ uint64_t shared_mem[];
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < array_length; 
         i += blockDim.x * gridDim.x)
    {
        printf("wow, %u %u \n", i, i+ array_length);
        shared_mem[i] = A[i];
        shared_mem[i+array_length] = B[i];

    }
    __syncthreads();
    int a;
    int b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int(shared_mem, i, number_length);
        b = decode_int(shared_mem, i+elementcount,number_length);
        printf("i, %u %u %u \n", i, a, b);
        add(C, a, b, number_length, i);

    }
}


__global__ 
void zero_sup_yes2(uint64_t* A, uint64_t* B, int number_length, int array_length,int elementcount,  uint64_t* C){
    extern __shared__ uint64_t shared_mem[];
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < array_length; 
         i += blockDim.x * gridDim.x)
    {
        shared_mem[i] = A[i];
        shared_mem[i+array_length] = B[i];
        shared_mem[i+2*array_length] = 0;

    }
    __syncthreads();
    int a;
    int b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int(shared_mem, i, number_length);
        b = decode_int(shared_mem, i+elementcount,number_length);
        compare(shared_mem, a, b, number_length, i+2*elementcount);

    }
    __syncthreads();
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < array_length; 
         i += blockDim.x * gridDim.x)
    {
        C[i] = shared_mem[i+2*array_length];

    }
}


struct Slabs{
    int array_length;
    int number_length;
    uint64_t* array;
};


Slabs encode(std::vector<std::string> vector){
    std::vector<uint64_t> helper_array;
    int length = vector.at(0).size();
    int size = 64 / length;
    uint64_t slab = 0;
    uint64_t helper = 0;
    int position = 0;
    Slabs s;
    s.number_length = length;

    for(std::string number:vector){
        helper = (uint64_t) (std::bitset<64>(number)).to_ulong();
        helper = helper << (position*length);
        slab = slab + helper;
        position++;
        if(position == size){
            position = 0;
            helper_array.push_back(slab);
            slab = 0;
        }
    }
    if(slab != 0){
       helper_array.push_back(slab); 
    }

    uint64_t* answer = (uint64_t*)malloc(helper_array.size() * sizeof(uint64_t));
    for(int i=0;i< helper_array.size();i++){
        answer[i] = helper_array.at(i);
    }
    s.array_length = helper_array.size();
    s.array = answer;
    return s;

}


void removeLeadingZeros(std::vector<std::string> &vector)
{
    int length;
    int max = 0;
    for (auto i = 0; i < vector.size(); i++)
    {
        length = vector[i].size() - vector[i].find('1');
        if(length > max){
            max = length;
        }
    }
    if( max == 0){
        return;
    }
    for(auto i = 0; i < vector.size(); i++){
        if(vector[i].size() > max){
        vector[i].erase(0, vector[i].size() - max);
        }
        else{
            vector[i].insert(0, max - vector[i].size(), '0');
        }
    }
}

int main()
{

    std::vector<std::string> h{"00000110000100000000", "110010100000000", "0100000000", "110001100000000", "010101100000001", "010001000000000", "100000100000000", "010101100010000"};
    std::vector<std::string> h2{"0010000100000001", "0000010100000001", "0000010100000001", "0000001100000001", "0010101100000000", "0010001000000001", "0100000100000001", "110101100100001"};

    removeLeadingZeros(h);
    removeLeadingZeros(h2);

    for(auto i = 0; i < h.size(); i++){
        //std::cout<< h[i] << std::endl;
        std::cout<< h2[i] << std::endl;
    }

    
    Slabs s = encode(h);
    Slabs s2 = encode(h2);
    for(int i=0;i<s.array_length;i++){
        std::cout<< std::bitset<64>(s.array[i]) << std::endl;
        std::cout<< std::bitset<64>(s2.array[i]) << std::endl;
    }


    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;


    uint64_t* h_out;

    size_t bytes = s.array_length * sizeof(uint64_t);
    size_t bytes2 = s.array_length * sizeof(unsigned long long int);
    std::cout<< "wow" <<std::endl;
    std::cout<< bytes <<std::endl;
    std::cout<< bytes2 <<std::endl;
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes2);

    hipMemcpy(d_A, s.array, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, s2.array, bytes, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, bytes2);

    //2*s.array_length*sizeof(uint64_t)
    //64, 1024, 3*s.array_length*sizeof(uint64_t)
    //zero_sup_no<<<64, 1024>>>(d_A, d_B, s.number_length, s.array_length,h.size(),  d_C);
    hello_world<<<1, 1>>>();
    hipDeviceReset();


    hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

    for(int i=0;i<s.array_length;i++){
        std::cout<< "hello" <<std::endl;
        std::cout<< std::bitset<64>(h_out[i]) << std::endl;
    }


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_out);
}