
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <algorithm>
#include <math.h>
#include <inttypes.h>
#include <fstream>

#if __CUDA_ARCH__ < 600
__device__ uint64_t myAtomicAdd(uint64_t* address, uint64_t val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        (uint64_t)(val +
                              (uint64_t) (assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return (old);
}
#endif

__device__ 
uint64_t decode_int(uint64_t* array, int i, int number_length)
{
    int amount = 64 / number_length;
    int chunk = i/amount;
    int position = i % amount;
    uint64_t slab = array[chunk];
    uint64_t test = 1U << (uint64_t) number_length;
    uint64_t mask = test - 1;
    mask = mask << (position*number_length);
    uint64_t answer = slab & mask;
    answer = answer >> (position*number_length);
    return answer;
    
}

__device__ 
uint64_t decode_int_shared(uint64_t* array, int i, int number_length, int elementcount, int array_length)
{
    int i_real = i;
    int amount = 64 / number_length;
    int chunk = i/amount;
    int position = i % amount;
    if(i >= elementcount){
        i_real = i -elementcount;
        chunk = i_real/amount;
        chunk = chunk + array_length;
        position = i_real % amount;
    }
    uint64_t slab = array[chunk];
    //uint64_t slab = 0;
    uint64_t test = 1U << (uint64_t) number_length;
    uint64_t mask = test - 1;
    mask = mask << (position*number_length);
    uint64_t answer = slab & mask;
    answer = answer >> (position*number_length);
    
    return answer;
    
}

__device__ 
int getLength(uint64_t bits ){

    int size = 0;

    for (; bits != 0; bits >>= 1){

    size++;
    }

    return size;
}

__device__
void add_shared( uint64_t* C, uint64_t a, uint64_t b, int number_length, int i, int elementcount, int array_length){

    uint64_t c = a +b;

    //printf("wow3, %u %lld\n", i,(unsigned long long int) c);

    if(getLength(c)<=number_length){

        int amount = 64 / number_length;


        int i_real = i - 2*elementcount;
        int chunk = i_real/amount;
        chunk = chunk + array_length*2;
        int position = i_real % amount;

        c = c << (position*number_length);
        printf("wow3, %u %lld\n", i,(unsigned long long int) c);

        myAtomicAdd(&C[chunk], c);
    }

}

__device__
void add( uint64_t* C, uint64_t a, uint64_t b, int number_length, int i){

    uint64_t c = a +b;

    if(getLength(c)<=number_length){

        int amount = 64 / number_length;

        int chunk = i/amount;

        int position = i % amount;

        c = c << (position*number_length);

        myAtomicAdd(&C[chunk], c);
    }

}


__device__
void compare_shared(uint64_t* C, int a, int b, int number_length, int i, int elementcount, int array_length){
    if(a > b){
        uint64_t helper = 0;
        int amount = 64 / number_length;

        int i_real = i - 2*elementcount;
        int chunk = i_real/amount;
        chunk = chunk + array_length*2;
        int position = i_real % amount;
        helper = 1;
        helper = helper << (position*number_length);
        myAtomicAdd(&C[chunk],helper);
    }
}

__device__
void compare(uint64_t* C, int a, int b, int number_length, int i){
    if(a > b){
        uint64_t helper = 0;
        int amount = 64 / number_length;
        int chunk = i/amount;
        int position = i % amount;
        helper = 1;
        helper = helper << (position*number_length);
        myAtomicAdd(&C[chunk],helper);
    }
}

__global__
void hello_world()
{
 printf("Hello World From GPU!\n");
}

__global__
void rand_gpu(uint64_t* A, uint64_t* B , int n, int elementcount)
{
 printf("Hello World From GPU!\n");
}

__global__ 
void zero_sup_no(uint64_t* A, uint64_t* B, int number_length_A,int number_length_B, int array_length,int elementcount,  uint64_t* C){
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        
        a = decode_int(A, i, number_length_A);
        b = decode_int(B, i,number_length_B);

        add(C, a, b, number_length_A, i);



    }
}

__global__ 
void no_zero(uint64_t* A, uint64_t* B,int number_length_A, int elementcount, uint64_t* C){
    uint64_t c;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < elementcount; 
         i += blockDim.x * gridDim.x)
    {
        
        c = A[i] + B[i];
        
        if(getLength(c) <= number_length_A){
            C[i] = c;
        }
        else{
            C[i] = 0;
        }/**
        if(A[i] > B[i]){
            C[i] = 1;
        }
        else{
            C[i] = 0;
        }
            **/

    }
}

//3250, 6500
__global__ 
void zero_sup_yes(uint64_t* A, uint64_t* B, int number_length, int array_length,int elementcount, uint64_t* C){
    extern __shared__ uint64_t shared_mem[];
    int n = (array_length + (1500 - 1)) / 1500;
    int amount = 64 / number_length;
    //int n = array_length / 1500;
    int length = 1500;
    int ele = amount*1500;
    //printf("wow, %u \n", n);
    for(int j = 0;j<n-1;j++){
        

       // printf("wow4,%u %u %u %u %u\n",j, n, length, ele, elementcount);
    for (int i = threadIdx.x; i < length; i += blockDim.x )
    {
        //if(i==0){
          //  printf("wow3, %u %lld %lld\n", i,(unsigned long long int) A[i], (unsigned long long int) B[i]);
        //}
        //printf("wow, %u %u %u\n", i, i+ array_length, array_length);
       // printf("wow, %u %u %u\n", i, A[i], B[i]);
         //printf("wow, %lld %lld\n", (unsigned long long) A[i], (unsigned long long) B[i]);
        //printf("wow2, %u %u %u\n", i, elementcount, array_length);
        shared_mem[i] = A[i+length*j];
        shared_mem[i+length] = B[i+length*j];
        //shared_mem[10000] = 3;
    }
   
   //printf("wow3,%u %u %u %u %u\n",j, n, length, ele, elementcount);
    
    __syncthreads();
    //printf("wow4, %u %u %u %u\n", n, length, ele, elementcount);
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < ele; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int_shared(shared_mem, i, number_length, ele, length);
        b = decode_int_shared(shared_mem, i+ele,number_length, ele, length);
        //printf("wow2, %lld %lld\n", a, b);
        add(C, a, b, number_length, i+ele*j);

    }
    __syncthreads();
    }
    __syncthreads();
    length = array_length - (n-1)*1500;
    ele = elementcount - amount*1500*(n-1);
    //printf("wow4,%u %u %u %u %u\n",array_length, n, length, ele, elementcount);
    for (int i = threadIdx.x; i < length; i += blockDim.x )
    {
        //if(i==0){
          //  printf("wow3, %u %lld %lld\n", i,(unsigned long long int) A[i], (unsigned long long int) B[i]);
        //}
        //printf("wow, %u %u %u\n", i, i+ array_length, array_length);
       // printf("wow, %u %u %u\n", i, A[i], B[i]);
         //printf("wow, %lld %lld\n", (unsigned long long) A[i], (unsigned long long) B[i]);
        //printf("wow2, %u %u %u\n", i, elementcount, array_length);
        shared_mem[i] = A[i+1500*(n-1)];
        shared_mem[i+length] = B[i+1500*(n-1)];
        //shared_mem[10000] = 3;
    }
   
   //printf("wow3, %u %u %u %u\n", n, length, ele, elementcount);
    
    __syncthreads();
    //printf("wow4, %u %u %u %u\n", n, length, ele, elementcount);
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < ele; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int_shared(shared_mem, i, number_length, ele, length);
        b = decode_int_shared(shared_mem, i+ele,number_length, ele, length);
        //printf("wow2, %lld %lld\n", a, b);
        add(C, a, b, number_length, i+amount*1500*(n-1));

    }
    __syncthreads();
    

}

//252, 65
__global__ 
void zero_sup_yes2(uint64_t* A, uint64_t* B, int number_length, int array_length,int elementcount,  uint64_t* C){
    extern __shared__ uint64_t shared_mem[];

    int n = (array_length + (63 - 1)) / 63;
    int amount = 64 / number_length;
    //int n = array_length / 1500;
    int length = 63;
    int ele = amount*63;

    for(int j = 0;j<n-1;j++){

    for (int i = threadIdx.x; i < length; i += blockDim.x)
    {
        shared_mem[i] = A[i+length*j];
        shared_mem[i+length] = B[i+length*j];
        shared_mem[i+2*length] = 0;

    }
    __syncthreads();
    uint64_t a;
    uint64_t b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < ele; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int_shared(shared_mem, i, number_length, ele, length);
        b = decode_int_shared(shared_mem, i+ele,number_length, ele, length);
        //printf("wow4, %u %u\n", i, i+2*ele);
        add_shared(shared_mem, a, b, number_length, i+2*ele,  ele, length);

    }
    printf("wow4,%u %u %u %u %u\n",j, n, length, ele, elementcount);
    __syncthreads();
    for (int i = threadIdx.x; i < length; i += blockDim.x)
    {
        C[i+length*j] = shared_mem[i+2*length];

    }
    __syncthreads();
    }

    __syncthreads();
    length = array_length - (n-1)*63;
    ele = elementcount - amount*63*(n-1);
    //printf("wow4,%u %u %u %u %u\n",5, n, length, ele, elementcount);

    for (int i = threadIdx.x; i < length; i += blockDim.x)
    {
        shared_mem[i] = A[i+63*(n-1)];
        shared_mem[i+length] = B[i+63*(n-1)];
        shared_mem[i+2*length] = 0;

    }
    __syncthreads();
    int a;
    int b;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < ele; 
         i += blockDim.x * gridDim.x)
    {
        a = decode_int_shared(shared_mem, i, number_length, ele, length);
        b = decode_int_shared(shared_mem, i+ele,number_length, ele, length);
        printf("wow3, %u %u\n", i,i+2*ele);
        add_shared(shared_mem, a, b, number_length, i+2*ele,  ele, length);

    }
    __syncthreads();
    for (int i = threadIdx.x; i < length; i += blockDim.x)
    {
        printf("wow4, %u %lld\n", i,shared_mem[i+2*length]);
        C[i+63*(n-1)] = shared_mem[i+2*length];

    }
    __syncthreads();
}

uint64_t getLength_cpu(uint64_t bits ){
    uint64_t size = 0;

    for (; bits != 0; bits >>= 1)
    size++;


    return size;
}

std::vector<uint64_t> add_cpu(std::vector<uint64_t> a, std::vector<uint64_t> b, int element_length){
    uint64_t c;
    std::vector<uint64_t> C;
    for(int i=0;i<a.size();i++){
        c = a.at(i) + b.at(i);
        //if(i == 1){
          //  std::cout<<"heute, hier: "<<a.at(i) <<std::endl;
            //std::cout<<"heute, hier: "<<b.at(i) <<std::endl;
        //}
        if(getLength_cpu(c) <= element_length){
            C.push_back(c);
        }
        else{
            C.push_back(0);
        }
    }
    return C;
}

std::vector<uint64_t> compare_cpu(std::vector<uint64_t> a, std::vector<uint64_t> b){
    std::vector<uint64_t> C;
    for(int i=0;i<a.size();i++){
        //if(i == 0){
            //std::cout<<"heute, hier: "<<a.at(i) <<std::endl;
          //  std::cout<<"heute, hier: "<<b.at(i) <<std::endl;
        //}
        if(a.at(i) > b.at(i)){
            C.push_back(1);
        }
        else{
            C.push_back(0);
        }
    }
    return C;
}

std::vector<std::string> int_to_string(std::vector<uint64_t> v){
    std::vector<std::string> s;
    for(int i=0;i<v.size();i++){
        s.push_back(std::bitset< 64 >(v.at(i)).to_string());
    }
    return s;
}

std::vector<uint64_t> string_to_int(std::vector<std::string> s){
    uint64_t helper;
    std::vector<uint64_t> v;
    for(int i=0;i<s.size();i++){
        helper = (uint64_t) std::bitset< 64 >(s.at(i)).to_ulong();
        v.push_back(helper);
        
    }
    return v;
}


struct Slabs{
    int array_length;
    int number_length;
    uint64_t* array;
};


Slabs encode(std::vector<std::string> vector){
    std::vector<uint64_t> helper_array;
    int length = vector.at(1).size();
    int size = 64 / length;
    uint64_t slab = 0;
    uint64_t helper = 0;
    int position = 0;
    Slabs s;
    s.number_length = length;
    for(std::string number:vector){
        helper = (uint64_t) (std::bitset<64>(number)).to_ulong();
        helper = helper << (position*length);
        slab = slab + helper;
        position++;
        if(position == size){
            position = 0;
            helper_array.push_back(slab);
            slab = 0;
        }
    }
    if(slab != 0){
       helper_array.push_back(slab); 
    }

    uint64_t* answer = (uint64_t*)malloc(helper_array.size() * sizeof(uint64_t));
    for(int i=0;i< helper_array.size();i++){
        answer[i] = helper_array.at(i);
    }
    s.array_length = helper_array.size();
    s.array = answer;
    return s;

}

std::vector<uint64_t> decode(uint64_t* numbers, int number_length, int array_length){
    int amount = 64 / number_length;
    uint64_t mask = pow(2,number_length) -1;
    uint64_t slab;
    uint64_t answer;
    std::vector<uint64_t> decoded_numbers;
    for(int i =0;i<array_length;i++){
        slab = numbers[i];
        for(int j =0;j<amount;j++){
            mask = pow(2,number_length) -1;
            mask = mask << (j*number_length);
            answer = slab & mask;
            answer = answer >> (j*number_length);
            decoded_numbers.push_back(answer);
        }
    }
    return decoded_numbers;
}


void removeLeadingZeros(std::vector<std::string> &vector)
{
    int length;
    int max = 0;
    for (auto i = 0; i < vector.size(); i++)
    {
        length = vector[i].size() - vector[i].find('1');
        if(length > max){
            max = length;
        }
    }
    if( max == 0){
        return;
    }
    for(auto i = 0; i < vector.size(); i++){
        if(vector[i].size() > max){
        vector[i].erase(0, vector[i].size() - max);
        }
        else{
            vector[i].insert(0, max - vector[i].size(), '0');
        }
    }
}

void validate(std::vector<uint64_t> h, std::vector<uint64_t> d) {
    for (size_t i = 0; i < h.size(); i++) {
        if (h.at(i) != d.at(i)) {
            
           std::cout << "found invalidated field in element " << i << std::endl;
           std::cout << "on CPU side: " << h.at(i) << std::endl;
           std::cout << "on GPU side: " << d.at(i) << std::endl;
           /**
           std::cout << "on CPU side: " << j << std::endl;
           std::cout << "on GPU side: " << k << std::endl;
           std::cout << "on GPU side: " << l << std::endl;
            */
            
        }
    }
}

void generate(std::vector<uint64_t> &a, std::vector<uint64_t> &b , int n, int elementcount){
    int number = pow(2,n) -1;
    for(int i=0;i<elementcount;i++){
        a.push_back((uint64_t) rand() % number + 1);
        b.push_back((uint64_t) rand() % number + 1);
    }
}

void test(){
    //size_t elementcount=1048576;
    //96000
    //1000
    //size_t elementcount=134217728;
    //67108864
    //size_t elementcount=1000000;
    size_t elementcount=6500;
    //int length=8;
    //std::ofstream myFile("no_shared.csv");
    std::ofstream myFile("shared2_add2.csv");
    myFile << "kernel;element_count;bit_count;block_count;thread_count;time_ms;throughput\n";

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int l = 1; l<=64;l++){
        std::cout<< "l: "<<l<<std::endl;
    std::vector<uint64_t> a;
    std::vector<uint64_t> b;

    generate(a, b, l, elementcount);

    std::vector<std::string> h = int_to_string(a);
    std::vector<std::string> h2 = int_to_string(b);

    removeLeadingZeros(h);
    removeLeadingZeros(h2);

    
    Slabs s = encode(h);
    Slabs s2 = encode(h2);


    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;


    uint64_t* h_out;

    size_t bytes = s.array_length * sizeof(uint64_t);
    size_t bytes2 = s.array_length * sizeof(unsigned long long int);
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes2);

    hipMemcpy(d_A, s.array, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, s2.array, bytes, hipMemcpyHostToDevice);
    

    
    for(int i=16;i<=1024;i=i*2){
        std::cout<< "i: "<<i<<std::endl;
        for(int j=8;j<=512;j=j*2){
            std::cout<<"j: "<< j<<std::endl;
            hipMemset(d_C, 0, bytes2);

            hipFuncSetCacheConfig(reinterpret_cast<const void*>(zero_sup_yes), hipFuncCachePreferShared);
            
            //zero_sup_no<<<j, i >>>(d_A, d_B, s.number_length,h.size(),  d_C);
            //zero_sup_no<<<j, i>>>(d_A, d_B, s.number_length,s.number_length, s.array_length,h.size(),  d_C);
            if(3*s.array_length*sizeof(uint64_t) > 1512){
                hipEventRecord(start);
            zero_sup_yes2<<<j, i, 1512>>>(d_A, d_B, s.number_length, s.array_length,h.size(),  d_C);
            hipEventRecord(stop);
            }
            else{
            hipEventRecord(start);
            zero_sup_yes2<<<j, i, 3*s.array_length*sizeof(uint64_t)>>>(d_A, d_B, s.number_length, s.array_length,h.size(),  d_C);
            hipEventRecord(stop);
            }
            //zero_sup_yes<<<j, i, 2*s.array_length*sizeof(uint64_t)>>>(d_A, d_B, s.number_length, s.array_length,h.size(),  d_C);
            
    
            hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            int max_number_length;
            int max_array_length;
            if(s.number_length> s2.number_length){
            max_number_length = s.number_length;
            max_array_length = s.array_length;
        }
        else{
            max_number_length = s2.number_length;
            max_array_length = s2.array_length;
        }

        std::vector<uint64_t> decoded_numbers = decode(h_out, max_number_length, max_array_length);
        std::vector<uint64_t> c = add_cpu(a,b, max_number_length);
        //std::vector<uint64_t> c = compare_cpu(a,b);

        //validate(c, decoded_numbers, j, i, l);

            myFile << "no_shared2" << ";";
            myFile << elementcount << ";";
            myFile << l << ";";
            myFile << j << ";";
            myFile << i << ";";
            myFile << milliseconds << ";";
            myFile << s.array_length*8*2/milliseconds/1e6 << "\n";
        }
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_out);
    }

    
}

void test2(){
    //size_t elementcount=1048576;
    //96000
    //1000
    size_t elementcount=1000000;
    //int length=8;
    //std::ofstream myFile("no_shared.csv");
    std::ofstream myFile("no_zero_add3.csv");
    myFile << "kernel;element_count;bit_count;block_count;thread_count;time_ms;throughput\n";

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int l = 1; l<=64;l++){
        std::cout<< "l: "<<l<<std::endl;
    std::vector<uint64_t> a;
    std::vector<uint64_t> b;

    generate(a, b, l, elementcount);
    size_t bytes = elementcount * sizeof(uint64_t);
    size_t bytes2 = elementcount * sizeof(unsigned long long int);

    std::vector<std::string> h = int_to_string(a);
    std::vector<std::string> h2 = int_to_string(b);

    removeLeadingZeros(h);
    removeLeadingZeros(h2);

    
    Slabs s = encode(h);
    Slabs s2 = encode(h2);
    uint64_t* h_A;
    uint64_t* h_B;

    h_A = (uint64_t*)malloc(bytes);
    h_B = (uint64_t*)malloc(bytes);
    for(int i = 0;i<a.size();i++){
        h_A[i] = a.at(i);
        h_B[i] = b.at(i);
    }


    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;


    uint64_t* h_out;

    
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes2);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    

    
    for(int i=16;i<=1024;i=i*2){
        std::cout<< "i: "<<i<<std::endl;
        for(int j=8;j<=512;j=j*2){
            std::cout<<"j: "<< j<<std::endl;
            hipMemset(d_C, 0, bytes2);
	
            hipEventRecord(start);
            no_zero<<<j, i >>>(d_A, d_B, s.number_length,h.size(),  d_C);
            hipEventRecord(stop);
    
            hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            int max_number_length;
            //int max_array_length;
            if(s.number_length> s2.number_length){
            max_number_length = s.number_length;
            //max_array_length = s.array_length;
        }
        else{
            max_number_length = s2.number_length;
            //max_array_length = s2.array_length;
        }

        //std::vector<uint64_t> decoded_numbers = decode(h_out, max_number_length, max_array_length);
        std::vector<uint64_t> c = add_cpu(a,b, max_number_length);
        //std::vector<uint64_t> c = add_cpu(a,b);
        std::vector<uint64_t> decoded_numbers;
        for(int s=0;s<a.size();s++){
            decoded_numbers.push_back(h_out[s]);
        }

        //validate(c, decoded_numbers);

            myFile << "no_shared" << ";";
            myFile << elementcount << ";";
            myFile << l << ";";
            myFile << j << ";";
            myFile << i << ";";
            myFile << milliseconds << ";";
            myFile << elementcount*8*2/milliseconds/1e6 << "\n";
        }
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_out);
    }

    
}

int main()
{   
    //test();
    
    std::vector<uint64_t> a;
    std::vector<uint64_t> b;

    generate(a, b,2, 20);
    for(uint64_t i:a)
        std::cout<<i<<", ";

    std::cout<<std::endl;

    for(uint64_t i:b)
        std::cout<<i<<", ";
    //std::vector<std::string> h{"00000110000100000000", "110010100000000", "1100000000", "110001100000000", "110101100000001", "110001000000000", "100000100000000", "110101100010000"};
    //std::vector<std::string> h2{"001000100000001", "10010100000001", "10010100000001", "10001100000001", "10101100000000", "10001000000001", "100000100000001", "110101100100001"};
    std::vector<std::string> h = int_to_string(a);
    std::vector<std::string> h2 = int_to_string(b);

    removeLeadingZeros(h);
    removeLeadingZeros(h2);

    //std::vector<uint64_t> a = string_to_int(h);
    //std::vector<uint64_t> b = string_to_int(h2);
    
    Slabs s = encode(h);
    Slabs s2 = encode(h2);


    uint64_t* d_A;
    uint64_t* d_B;
    uint64_t* d_C;


    uint64_t* h_out;

    size_t bytes = s.array_length * sizeof(uint64_t);
    size_t bytes2 = s.array_length * sizeof(unsigned long long int);

    std::cout << bytes<<std::endl;
        std::cout<< bytes2<<std::endl;
    h_out = (uint64_t*)malloc(bytes);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes2);

    hipMemcpy(d_A, s.array, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, s2.array, bytes, hipMemcpyHostToDevice);
    //for(int i=0;i<s.array_length;i++){
      //  std::cout<< i <<", " << s.array[i]<<std::endl;
        //std::cout<< i <<", " << s2.array[i]<<std::endl;
    //}
    hipMemset(d_C, 0, bytes2);
    //2*s.array_length*sizeof(uint64_t)
    //64, 1024, 3*s.array_length*sizeof(uint64_t)
    std::cout<<"hello "<< 3*s.array_length*sizeof(uint64_t)<<std::endl;
    std::cout<<"hello "<< sizeof(uint64_t)<<std::endl;
    //cudaFuncSetAttribute(zero_sup_yes2, cudaFuncAttributeMaxDynamicSharedMemorySize, 65536);
   // if(2*s.array_length*sizeof(uint64_t) > 24000){
   //     zero_sup_yes<<<64, 1024, 24000>>>(d_A, d_B, s.number_length, s.array_length,h.size(),  d_C);
   // }
   // else{
    //    std::cout<<"hier"<<std::endl;
    zero_sup_yes2<<<256, 16, 3*s.array_length*sizeof(uint64_t)>>>(d_A, d_B, s.number_length, s.array_length,h.size(),  d_C);
    //}
    
    //zero_sup_no<<<64, 1024>>>(d_A, d_B, s.number_length,s.number_length, s.array_length,h.size(),  d_C);
    //hello_world<<<1, 1>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));


    hipMemcpy(h_out, d_C, bytes, hipMemcpyDeviceToHost);

    int max_number_length;
    int max_array_length;
    if(s.number_length> s2.number_length){
        max_number_length = s.number_length;
        max_array_length = s.array_length;
    }
    else{
        max_number_length = s2.number_length;
        max_array_length = s2.array_length;
    }
    std::vector<uint64_t> decoded_numbers = decode(h_out, max_number_length, max_array_length);

    std::vector<uint64_t> c = add_cpu(a,b, max_number_length);

    validate(c, decoded_numbers);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_out);
    
    
    
}