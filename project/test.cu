
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void hello_world()
{
 printf("Hello World From GPU!\n");
}

int main(void)
{
  hello_world<<<1, 1>>>();
  hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
  return 0;
}
